#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <cstdarg>
#include "pred.h"

/*Determines the maximum from a set of values*/
int maximo(int count, ...)
{
	va_list ap;
    	int j, temp, mx = 0;
    	va_start(ap, count);

	for(j = 0; j < count; j++)
	{
		temp = va_arg(ap, int);
		if(temp > mx)
			mx = temp;
	}

    	va_end(ap);
    	return mx;
}


__global__ void bpreds(int *dop1, int *dop2, int rows, int of1, int of2, int *cons, int numc, int nx, int *res, int *res2)
{
 	extern __shared__ int shared[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int x, rowact, rowact1, op1, op2;
	if(threadIdx.x < numc)
		shared[threadIdx.x] = cons[threadIdx.x];
	__syncthreads();
	if(id < rows)
	{
		rowact1 = id * of1;
		rowact = id * of2;
		for(x = nx; x < numc; x += 3)
		{
			op1 = shared[x+1];
			if(op1 < 0)
				op1 = dop1[rowact1 - op1 - 1];
			else
				op1 = dop2[rowact + op1];
			op2 = shared[x+2];
			if(op2 < 0)
				op2 = dop1[rowact1 - op2 - 1];
			else
				op2 = dop2[rowact + op2];
			switch(shared[x] - BPOFFSET)
			{
				case SBG_EQ: if(op1 != op2)
						return;
				break;
				case SBG_GT: if(op1 <= op2)
						return;
				break;
				case SBG_LT: if(op1 >= op2)
						return;
				break;
				case SBG_GE: if(op1 < op2)
						return;
				break;
				case SBG_LE: if(op1 > op2)
						return;
				break;
				case SBG_DF: if(op1 == op2)
						return;
			}
		}
		if(res2 != NULL)
			res2[id] = 1; 
		for(x = 0; x < nx; x += 3)
		{
			op1 = shared[x+1];
			if(op1 < 0)
				op1 *= -1;
			else
				op1 = dop2[rowact + op1];
			op2 = shared[x+2];
			if(op2 < 0)
				op2 *= -1;
			else
				op2 = dop2[rowact + op2];
			switch(shared[x])
			{
				case SBG_EQ: if(op1 != op2)
						return;
				break;
				case SBG_GT: if(op1 <= op2)
						return;
				break;
				case SBG_LT: if(op1 >= op2)
						return;
				break;
				case SBG_GE: if(op1 < op2)
						return;
				break;
				case SBG_LE: if(op1 > op2)
						return;
				break;
				case SBG_DF: if(op1 == op2)
						return;
			}
		}
		res[id] = 1;
	}
}

/*Mark all rows that comply with the comparison predicates*/
__global__ void bpredsnormal2(int *dop1, int rows, int of1, int *cons, int numc, int *res)
{
 	extern __shared__ int shared[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int x, rowact, op1, op2;
	if(threadIdx.x < numc)
		shared[threadIdx.x] = cons[threadIdx.x];
	__syncthreads();
	if(id < rows)
	{
		rowact = id * of1; 
		for(x = 0; x < numc; x += 3)
		{
			op1 = shared[x+1];
			if(op1 < 0)
				op1 *= -1;
			else
				op1 = dop1[rowact + op1];
			op2 = shared[x+2];
			if(op2 < 0)
				op2 *= -1;
			else
				op2 = dop1[rowact + op2];
			switch(shared[x])
			{
				case SBG_EQ: if(op1 != op2)
						return;
				break;
				case SBG_GT: if(op1 <= op2)
						return;
				break;
				case SBG_LT: if(op1 >= op2)
						return;
				break;
				case SBG_GE: if(op1 < op2)
						return;
				break;
				case SBG_LE: if(op1 > op2)
						return;
				break;
				case SBG_DF: if(op1 == op2)
						return;
			}
		}
		res[id] = 1;
	}
}

/*Unmark all rows that do not comply with the comparison predicates*/
__global__ void bpredsnormal(int *dop1, int rows, int of1, int *cons, int numc, int *res)
{
 	extern __shared__ int shared[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int x, rowact, op1, op2;
	if(threadIdx.x < numc)
		shared[threadIdx.x] = cons[threadIdx.x];
	__syncthreads();
	if(id < rows)
	{
		if(res[id] == 0)
			return;
		rowact = id * of1; 
		for(x = 0; x < numc; x += 3)
		{
			op1 = shared[x+1];
			if(op1 < 0)
				op1 *= -1;
			else
				op1 = dop1[rowact + op1];
			op2 = shared[x+2];
			if(op2 < 0)
				op2 *= -1;
			else
				op2 = dop1[rowact + op2];
			switch(shared[x])
			{
				case SBG_EQ: if(op1 != op2)
					     {
						res[id] = 0;
						return;
					     }
				break;
				case SBG_GT: if(op1 <= op2)
					     {
						res[id] = 0;
						return;
					     }
				break;
				case SBG_LT: if(op1 >= op2)
					     {
						res[id] = 0;
						return;
					     }
				break;
				case SBG_GE: if(op1 < op2)
					     {
						res[id] = 0;
						return;
					     }
				break;
				case SBG_LE: if(op1 > op2)
					     {
						res[id] = 0;
						return;
					     }
				break;
				case SBG_DF: if(op1 == op2)
					     {
						res[id] = 0;
						return;
					     }
			}
		}
	}
}

__global__ void bpredsOR(int *dop1, int *dop2, int rows, int of1, int of2, int *cons, int numc, int nx, int *res, int *res2)
{
 	extern __shared__ int shared[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int x, rowact, rowact1, op1, op2;
	if(threadIdx.x < numc)
		shared[threadIdx.x] = cons[threadIdx.x];
	__syncthreads();
	if(id < rows)
	{
		rowact1 = id * of1;
		rowact = id * of2;
		for(x = nx; x < numc; x += 3)
		{
			op1 = shared[x+1];
			if(op1 < 0)
				op1 = dop1[rowact1 - op1 - 1];
			else
				op1 = dop2[rowact + op1];
			op2 = shared[x+2];
			if(op2 < 0)
				op2 = dop1[rowact1 - op2 - 1];
			else
				op2 = dop2[rowact + op2];
			switch(shared[x] - BPOFFSET)
			{
				case SBG_EQ: if(op1 == op2)
					     {
						res2[id] = 1;
						x = numc;
					     }
				break;
				case SBG_GT: if(op1 > op2)
					     {
						res2[id] = 1;
						x = numc;
					     }
				break;
				case SBG_LT: if(op1 < op2)
					     {
						res2[id] = 1;
						x = numc;
					     }
				break;
				case SBG_GE: if(op1 >= op2)
					     {
						res2[id] = 1;
						x = numc;
					     }
				break;
				case SBG_LE: if(op1 <= op2)
					     {
						res2[id] = 1;
						x = numc;
					     }
				break;
				case SBG_DF: if(op1 != op2)
					     {
						res2[id] = 1;
						x = numc;
					     }
			}
		}
		for(x = 0; x < nx; x += 3)
		{
			op1 = shared[x+1];
			if(op1 < 0)
				op1 *= -1;
			else
				op1 = dop2[rowact + op1];
			op2 = shared[x+2];
			if(op2 < 0)
				op2 *= -1;
			else
				op2 = dop2[rowact + op2];
			switch(shared[x])
			{
				case SBG_EQ: if(op1 == op2)
					     {
						res[id] = 1;
						return;
					     }
				break;
				case SBG_GT: if(op1 > op2)
					     {
						res[id] = 1;
						return;
					     }
				break;
				case SBG_LT: if(op1 < op2)
					     {
						res[id] = 1;
						return;
					     }
				break;
				case SBG_GE: if(op1 >= op2)
					     {
						res[id] = 1;
						return;
					     }
				break;
				case SBG_LE: if(op1 <= op2)
					     {
						res[id] = 1;
						return;
					     }
				break;
				case SBG_DF: if(op1 != op2)
					     {
						res[id] = 1;
						return;
					     }
			}
		}
	}
}

/*Mark all rows that comply with the comparison predicates using disjunctions (i.e. a row is marked if it complies with at least one predicate)*/
__global__ void bpredsorlogic2(int *dop1, int rows, int of1, int *cons, int numc, int *res)
{
 	extern __shared__ int shared[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int x, rowact, op1, op2;
	if(threadIdx.x < numc)
		shared[threadIdx.x] = cons[threadIdx.x];
	__syncthreads();
	if(id < rows)
	{
		rowact = id * of1; 
		for(x = 0; x < numc; x += 3)
		{
			op1 = shared[x+1];
			if(op1 < 0)
				op1 *= -1;
			else
				op1 = dop1[rowact + op1];
			op2 = shared[x+2];
			if(op2 < 0)
				op2 *= -1;
			else
				op2 = dop1[rowact + op2];
			switch(shared[x])
			{
				case SBG_EQ: if(op1 == op2)
					     {
						res[id] = 1;
						return;
					     }
				break;
				case SBG_GT: if(op1 > op2)
					     {
						res[id] = 1;
						return;
					     }
				break;
				case SBG_LT: if(op1 < op2)
					     {
						res[id] = 1;
						return;
					     }
				break;
				case SBG_GE: if(op1 >= op2)
					     {
						res[id] = 1;
						return;
					     }
				break;
				case SBG_LE: if(op1 <= op2)
					     {
						res[id] = 1;
						return;
					     }
				break;
				case SBG_DF: if(op1 != op2)
					     {
						res[id] = 1;
						return;
					     }
			}
		}
		
	}
}

/*Unmark all rows that do not comply with the comparison predicates using disjunctions (i.e. a row is unmarked only if it complies with none of the predicates)*/
__global__ void bpredsorlogic(int *dop1, int rows, int of1, int *cons, int numc, int *res)
{
 	extern __shared__ int shared[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int x, rowact, op1, op2;
	if(threadIdx.x < numc)
		shared[threadIdx.x] = cons[threadIdx.x];
	__syncthreads();
	if(id < rows)
	{
		if(res[id] == 0)
			return;
		rowact = id * of1; 
		for(x = 0; x < numc; x += 3)
		{
			op1 = shared[x+1];
			if(op1 < 0)
				op1 *= -1;
			else
				op1 = dop1[rowact + op1];
			op2 = shared[x+2];
			if(op2 < 0)
				op2 *= -1;
			else
				op2 = dop1[rowact + op2];
			switch(shared[x])
			{
				case SBG_EQ: if(op1 == op2)
						return;
				break;
				case SBG_GT: if(op1 > op2)
						return;
				break;
				case SBG_LT: if(op1 < op2)
						return;
				break;
				case SBG_GE: if(op1 >= op2)
						return;
				break;
				case SBG_LE: if(op1 <= op2)
						return;
				break;
				case SBG_DF: if(op1 != op2)
						return;
			}
		}
		res[id] = 0;
	}
}

