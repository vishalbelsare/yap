#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <stdlib.h>
#include "memory.h"
#include "bpreds.h"

/*Mark all rows that comply with the selections*/
__global__ void marcar2(int *dop1, int rows, int cols, int *cons, int numc, int *res)
{
 	extern __shared__ int shared[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int x, rowact, posact;
	if(threadIdx.x < numc)
		shared[threadIdx.x] = cons[threadIdx.x];
	__syncthreads();
	if(id < rows)
	{
		rowact = id * cols;
		for(x = 0; x < numc; x += 2)
		{
			posact = rowact + shared[x];
			if(dop1[posact] != shared[x+1])
				return;
		}
		res[id] = 1;
	}
}
/*If we already have an array of marks (perhaps because the selfjoin was applied first), 
we unmark any rows that do not comply with the selections*/
__global__ void marcar(int *dop1, int rows, int cols, int *cons, int numc, int *res)
{
	extern __shared__ int shared[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int x, rowact, posact;
	if(threadIdx.x < numc)
		shared[threadIdx.x] = cons[threadIdx.x];
	__syncthreads();
	if(id < rows)
	{
		if(res[id] == 0)
			return;
		rowact = id * cols;
		for(x = 0; x < numc; x += 2)
		{
			posact = rowact + shared[x];
			if(dop1[posact] != shared[x+1])
			{
				res[id] = 0;
				return;
			}
		}
	}
}

/*Unmark all rows that do not comply with the selfjoins.*/
__global__ void samejoin(int *dop1, int rows, int cols, int *dhead, int cont, int *res)
{
	extern __shared__ int shared[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int temp, temp2, pos, x, y;
	if(threadIdx.x < cont)
		shared[threadIdx.x] = dhead[threadIdx.x];
	__syncthreads();
	if(id < rows)
	{	
		if(res[id] == 0)
			return;
		pos = id * cols;
		for(x = 0; x < cont; x++)
		{
			temp = dop1[pos+shared[x]];
			y = x + 1;
			temp2 = shared[y];
			while(temp2 > -1)
			{
				if(temp != dop1[temp2+pos])
				{
					res[id] = 0;
					return;
				}
				y++;
				temp2 = shared[y];
			}
			x = y;
		}
	}
}

/*Mark all rows that comply with the selfjoins*/
__global__ void samejoin2(int *dop1, int rows, int cols, int *dhead, int cont, int *res)
{
	extern __shared__ int shared[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int temp, temp2, pos, x, y;
	if(threadIdx.x < cont)
		shared[threadIdx.x] = dhead[threadIdx.x];
	__syncthreads();
	if(id < rows)
	{	
		pos = id * cols;
		for(x = 0; x < cont; x++)
		{
			temp = dop1[pos+shared[x]];
			y = x + 1;
			temp2 = shared[y];
			while(temp2 > -1)
			{
				if(temp != dop1[temp2+pos])
					return;
				y++;
				temp2 = shared[y];
			}
			x = y;
		}
		res[id] = 1;
	}
}

/*Project all columns found in 'dhead' to a new array 'res'*/
__global__ void proyectar(int *dop1, int rows, int cols, int *dhead, int hsize, int *res)
{
	extern __shared__ int shared[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int pos, posr, x;
	if(threadIdx.x < hsize)
		shared[threadIdx.x] = dhead[threadIdx.x];
	__syncthreads();
	if(id < rows)
	{	
		pos = id * cols;
		posr = id * hsize;
		for(x = 0; x < hsize; x++, posr++)
			res[posr] = dop1[pos+shared[x]];
	}
}

/*Project all columns found in 'dhead' using only the rows marked as valid (i.e. those that complied with 
selections, selfjoins, etc.). The array 'temp' holds the result of the prefix sum of said marks.*/
__global__ void llenarproyectar(int *dop1, int rows, int cols, int *temp, int *dhead, int hsize, int *res)
{
	extern __shared__ int shared[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int pos, posr, x;
	if(threadIdx.x < hsize)
		shared[threadIdx.x] = dhead[threadIdx.x];
	__syncthreads();
	if(id < rows)
	{		
		posr = temp[id];
		if(temp[id+1] != posr)
		{
			pos = id * cols;
			posr *= hsize;			
			for(x = 0; x < hsize; x++, posr++)
				res[posr] = dop1[pos+shared[x]];
		}
	}
}

/*Performs selections, selfjoins and comparison predicates when the rule has a single normal predicate.*/
int selectproyect(int *dop1, int rows, int cols, int head_size, int *select, int numselect, int *selfjoin, int numselfj, int *preds, int numpreds, int *project, int **ret, int ANDlogic)
{
	int *fres = NULL, *temp = NULL;
	int *dhead = NULL, tmplen;
	int size, size2, num;
	thrust::device_ptr<int> res;

#if TIMER
	cuda_stats.selects++;
#endif

	int head_bytes = maximo(4, numselect, numselfj, numpreds, head_size) * sizeof(int);
	reservar(&dhead, head_bytes);
	int numthreads = 1024;
	//int numthreads = 32;
	int blockllen = rows / numthreads + 1;

	#ifdef ROCKIT
		ANDlogic = 1;
	#endif

	if(numselect > 0)
	{		
		tmplen = rows + 1;
		size2 = tmplen * sizeof(int);
		reservar(&temp, size2);
		hipMemset(temp, 0, size2);
		size = numselect * sizeof(int);
		hipMemcpy(dhead, select, size, hipMemcpyHostToDevice);

		marcar2<<<blockllen, numthreads, size>>>(dop1, rows, cols, dhead, numselect, temp + 1);
		
		if(numselfj > 0)
		{
			size = numselfj * sizeof(int);
			hipMemcpy(dhead, selfjoin, size, hipMemcpyHostToDevice);
			samejoin<<<blockllen, numthreads, size>>>(dop1, rows, cols, dhead, numselfj, temp + 1);
		}

		if(numpreds > 0)
		{
			size = numpreds * sizeof(int);
			hipMemcpy(dhead, preds, size, hipMemcpyHostToDevice);
			if(ANDlogic)
				bpredsnormal<<<blockllen, numthreads, size>>>(dop1, rows, cols, dhead, numpreds, temp + 1);
			else
				bpredsorlogic<<<blockllen, numthreads, size>>>(dop1, rows, cols, dhead, numpreds, temp + 1);
		}

		res = thrust::device_pointer_cast(temp);
		thrust::inclusive_scan(res + 1, res + tmplen, res + 1);
		num = res[rows];
		if(num == 0)
			return 0;

		size = head_size * sizeof(int);
		reservar(&fres, num * size);
		hipMemcpy(dhead, project, size, hipMemcpyHostToDevice);
		llenarproyectar<<<blockllen, numthreads, size>>>(dop1, rows, cols, temp, dhead, head_size, fres);
		hipFree(dhead);
		hipFree(temp);
		*ret = fres;
		return num;
	}
	else
	{
		if(numselfj > 0)
		{
			tmplen = rows + 1;
			size2 = tmplen * sizeof(int);
			reservar(&temp, size2);
			hipMemset(temp, 0, size2);
			size = numselfj * sizeof(int);
			hipMemcpy(dhead, selfjoin, size, hipMemcpyHostToDevice);
			samejoin2<<<blockllen, numthreads, size>>>(dop1, rows, cols, dhead, numselfj, temp + 1);

			if(numpreds > 0)
			{
				size = numpreds * sizeof(int);
				hipMemcpy(dhead, preds, size, hipMemcpyHostToDevice);
				if(ANDlogic)
					bpredsnormal<<<blockllen, numthreads, size>>>(dop1, rows, cols, dhead, numpreds, temp + 1);
				else
					bpredsorlogic<<<blockllen, numthreads, size>>>(dop1, rows, cols, dhead, numpreds, temp + 1);

			}

			res = thrust::device_pointer_cast(temp);
			thrust::inclusive_scan(res + 1, res + tmplen, res + 1);
			num = res[rows];
			if(num == 0)
				return 0;

			size = head_size * sizeof(int);
			reservar(&fres, num * size);
			hipMemcpy(dhead, project, size, hipMemcpyHostToDevice);
			llenarproyectar<<<blockllen, numthreads, size>>>(dop1, rows, cols, temp, dhead, head_size, fres);
			hipFree(dhead);
			hipFree(temp);
			*ret = fres;
			return num;
		}
		else
		{
			if(numpreds > 0)
			{
				tmplen = rows + 1;
				size2 = tmplen * sizeof(int);
				reservar(&temp, size2);
				hipMemset(temp, 0, size2);		
				size = numpreds * sizeof(int);
				hipMemcpy(dhead, preds, size, hipMemcpyHostToDevice);

				if(ANDlogic)
					bpredsnormal2<<<blockllen, numthreads, size>>>(dop1, rows, cols, dhead, numpreds, temp + 1);					
				else
					bpredsorlogic2<<<blockllen, numthreads, size>>>(dop1, rows, cols, dhead, numpreds, temp + 1);
				res = thrust::device_pointer_cast(temp);
				thrust::inclusive_scan(res + 1, res + tmplen, res + 1);
				num = res[rows];

				if(num == 0)
					return 0;

				size = head_size * sizeof(int);
				reservar(&fres, num * size);
				hipMemcpy(dhead, project, size, hipMemcpyHostToDevice);
				llenarproyectar<<<blockllen, numthreads, size>>>(dop1, rows, cols, temp, dhead, head_size, fres);
				hipFree(dhead);
				hipFree(temp);
				*ret = fres;
				return num;
			}
			else
			{
				size = head_size * sizeof(int);
				reservar(&fres, rows * size);
				hipMemcpy(dhead, project, size, hipMemcpyHostToDevice);
				proyectar<<<blockllen, numthreads, size>>>(dop1, rows, cols, dhead, head_size, fres);
				hipFree(dhead);
				*ret = fres;
				return rows;
			}
		}
	}
}
