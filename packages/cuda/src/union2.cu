/*Computer generated file to remove duplicates. Since Thrust's unique and sort, unlike their std's counterparts, don't have a way to specify the size of each element in
the array, comparing pairs, triplets and other sets is not possible without defining a new pointer and all related operations for each set. If you have a better idea to do
this, please don't hesitate to email us.*/

#include <thrust/device_vector.h>
#include <thrust/unique.h>
#include <thrust/distance.h>
#include <thrust/sort.h>
#include <iostream>
#include "memory.h"
#include "union2.h"

int unir(int *res, int rows, int tipo, int **ret, int final)
{
	thrust::device_ptr<int> pt, re;
	thrust::device_ptr<s2> pt2, re2;
	thrust::device_ptr<s3> pt3, re3;
	thrust::device_ptr<s4> pt4, re4;
	thrust::device_ptr<s5> pt5, re5;
	thrust::device_ptr<s6> pt6, re6;
	thrust::device_ptr<s7> pt7, re7;
	thrust::device_ptr<s8> pt8, re8;
	thrust::device_ptr<s9> pt9, re9;
	thrust::device_ptr<s10> pt10, re10;
	thrust::device_ptr<s11> pt11, re11;
	thrust::device_ptr<s12> pt12, re12;
	thrust::device_ptr<s13> pt13, re13;
	thrust::device_ptr<s14> pt14, re14;
	thrust::device_ptr<s15> pt15, re15;
	thrust::device_ptr<s16> pt16, re16;
	thrust::device_ptr<s17> pt17, re17;
	thrust::device_ptr<s18> pt18, re18;
	thrust::device_ptr<s19> pt19, re19;
	thrust::device_ptr<s20> pt20, re20;
	s2 *t2;
	s3 *t3;
	s4 *t4;
	s5 *t5;
	s6 *t6;
	s7 *t7;
	s8 *t8;
	s9 *t9;
	s10 *t10;
	s11 *t11;
	s12 *t12;
	s13 *t13;
	s14 *t14;
	s15 *t15;
	s16 *t16;
	s17 *t17;
	s18 *t18;
	s19 *t19;
	s20 *t20;
	int flag, nrows, *nres, size;

#if TIMER
	cuda_stats.unions++;
#endif

	switch(tipo)
	{
		case 1:
		{
			pt = thrust::device_pointer_cast(res);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt, pt + rows);
					if(final)
					{
						re = thrust::unique(pt, pt + rows, q1());
						re = thrust::unique(pt, re);
					}
					else
						re = thrust::unique(pt, pt + rows);
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt, re);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
		case 2:
		{
			t2 = (s2*)res;
			pt2 = thrust::device_pointer_cast(t2);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt2, pt2 + rows, o2());
					if(final)
					{
						re2 = thrust::unique(pt2, pt2 + rows, q2());
						re2 = thrust::unique(pt2, re2, p2());
					}
					else
						re2 = thrust::unique(pt2, pt2 + rows, p2());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt2, re2);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
		case 3:
		{
			t3 = (s3*)res;
			pt3 = thrust::device_pointer_cast(t3);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt3, pt3 + rows, o3());
					if(final)
					{
						re3 = thrust::unique(pt3, pt3 + rows, q3());
						re3 = thrust::unique(pt3, re3, p3());
					}
					else
						re3 = thrust::unique(pt3, pt3 + rows, p3());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt3, re3);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
		case 4:
		{
			t4 = (s4*)res;
			pt4 = thrust::device_pointer_cast(t4);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt4, pt4 + rows, o4());
					if(final)
					{
						re4 = thrust::unique(pt4, pt4 + rows, q4());
						re4 = thrust::unique(pt4, re4, p4());
					}
					else
						re4 = thrust::unique(pt4, pt4 + rows, p4());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt4, re4);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
		case 5:
		{
			t5 = (s5*)res;
			pt5 = thrust::device_pointer_cast(t5);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt5, pt5 + rows, o5());
					if(final)
					{
						re5 = thrust::unique(pt5, pt5 + rows, q5());
						re5 = thrust::unique(pt5, re5, p5());
					}
					else
						re5 = thrust::unique(pt5, pt5 + rows, p5());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt5, re5);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
		case 6:
		{
			t6 = (s6*)res;
			pt6 = thrust::device_pointer_cast(t6);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt6, pt6 + rows, o6());
					if(final)
					{
						re6 = thrust::unique(pt6, pt6 + rows, q6());
						re6 = thrust::unique(pt6, re6, p6());
					}
					else
						re6 = thrust::unique(pt6, pt6 + rows, p6());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt6, re6);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
		case 7:
		{
			t7 = (s7*)res;
			pt7 = thrust::device_pointer_cast(t7);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt7, pt7 + rows, o7());
					if(final)
					{
						re7 = thrust::unique(pt7, pt7 + rows, q7());
						re7 = thrust::unique(pt7, re7, p7());
					}
					else
						re7 = thrust::unique(pt7, pt7 + rows, p7());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt7, re7);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
		case 8:
		{
			t8 = (s8*)res;
			pt8 = thrust::device_pointer_cast(t8);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt8, pt8 + rows, o8());
					if(final)
					{
						re8 = thrust::unique(pt8, pt8 + rows, q8());
						re8 = thrust::unique(pt8, re8, p8());
					}
					else
						re8 = thrust::unique(pt8, pt8 + rows, p8());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt8, re8);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
		case 9:
		{
			t9 = (s9*)res;
			pt9 = thrust::device_pointer_cast(t9);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt9, pt9 + rows, o9());
					if(final)
					{
						re9 = thrust::unique(pt9, pt9 + rows, q9());
						re9 = thrust::unique(pt9, re9, p9());
					}
					else
						re9 = thrust::unique(pt9, pt9 + rows, p9());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt9, re9);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
		case 10:
		{
			t10 = (s10*)res;
			pt10 = thrust::device_pointer_cast(t10);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt10, pt10 + rows, o10());
					if(final)
					{
						re10 = thrust::unique(pt10, pt10 + rows, q10());
						re10 = thrust::unique(pt10, re10, p10());
					}
					else
						re10 = thrust::unique(pt10, pt10 + rows, p10());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt10, re10);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
		case 11:
		{
			t11 = (s11*)res;
			pt11 = thrust::device_pointer_cast(t11);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt11, pt11 + rows, o11());
					if(final)
					{
						re11 = thrust::unique(pt11, pt11 + rows, q11());
						re11 = thrust::unique(pt11, re11, p11());
					}
					else
						re11 = thrust::unique(pt11, pt11 + rows, p11());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt11, re11);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
		case 12:
		{
			t12 = (s12*)res;
			pt12 = thrust::device_pointer_cast(t12);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt12, pt12 + rows, o12());
					if(final)
					{
						re12 = thrust::unique(pt12, pt12 + rows, q12());
						re12 = thrust::unique(pt12, re12, p12());
					}
					else
						re12 = thrust::unique(pt12, pt12 + rows, p12());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt12, re12);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
		case 13:
		{
			t13 = (s13*)res;
			pt13 = thrust::device_pointer_cast(t13);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt13, pt13 + rows, o13());
					if(final)
					{
						re13 = thrust::unique(pt13, pt13 + rows, q13());
						re13 = thrust::unique(pt13, re13, p13());
					}
					else
						re13 = thrust::unique(pt13, pt13 + rows, p13());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt13, re13);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
		case 14:
		{
			t14 = (s14*)res;
			pt14 = thrust::device_pointer_cast(t14);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt14, pt14 + rows, o14());
					if(final)
					{
						re14 = thrust::unique(pt14, pt14 + rows, q14());
						re14 = thrust::unique(pt14, re14, p14());
					}
					else
						re14 = thrust::unique(pt14, pt14 + rows, p14());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt14, re14);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
		case 15:
		{
			t15 = (s15*)res;
			pt15 = thrust::device_pointer_cast(t15);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt15, pt15 + rows, o15());
					if(final)
					{
						re15 = thrust::unique(pt15, pt15 + rows, q15());
						re15 = thrust::unique(pt15, re15, p15());
					}
					else
						re15 = thrust::unique(pt15, pt15 + rows, p15());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt15, re15);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
		case 16:
		{
			t16 = (s16*)res;
			pt16 = thrust::device_pointer_cast(t16);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt16, pt16 + rows, o16());
					if(final)
					{
						re16 = thrust::unique(pt16, pt16 + rows, q16());
						re16 = thrust::unique(pt16, re16, p16());
					}
					else
						re16 = thrust::unique(pt16, pt16 + rows, p16());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt16, re16);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
		case 17:
		{
			t17 = (s17*)res;
			pt17 = thrust::device_pointer_cast(t17);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt17, pt17 + rows, o17());
					if(final)
					{
						re17 = thrust::unique(pt17, pt17 + rows, q17());
						re17 = thrust::unique(pt17, re17, p17());
					}
					else
						re17 = thrust::unique(pt17, pt17 + rows, p17());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt17, re17);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
		case 18:
		{
			t18 = (s18*)res;
			pt18 = thrust::device_pointer_cast(t18);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt18, pt18 + rows, o18());
					if(final)
					{
						re18 = thrust::unique(pt18, pt18 + rows, q18());
						re18 = thrust::unique(pt18, re18, p18());
					}
					else
						re18 = thrust::unique(pt18, pt18 + rows, p18());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt18, re18);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
		case 19:
		{
			t19 = (s19*)res;
			pt19 = thrust::device_pointer_cast(t19);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt19, pt19 + rows, o19());
					if(final)
					{
						re19 = thrust::unique(pt19, pt19 + rows, q19());
						re19 = thrust::unique(pt19, re19, p19());
					}
					else
						re19 = thrust::unique(pt19, pt19 + rows, p19());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt19, re19);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
		case 20:
		{
			t20 = (s20*)res;
			pt20 = thrust::device_pointer_cast(t20);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt20, pt20 + rows, o20());
					if(final)
					{
						re20 = thrust::unique(pt20, pt20 + rows, q20());
						re20 = thrust::unique(pt20, re20, p20());
					}
					else
						re20 = thrust::unique(pt20, pt20 + rows, p20());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}
			}
			nrows = thrust::distance(pt20, re20);
			if(nrows < rows / 2)
			{
				size = nrows * tipo * sizeof(int);
				reservar(&nres, size);
				hipMemcpyAsync(nres, res, size, hipMemcpyDeviceToDevice);
				hipFree(*ret);
				*ret = nres;
			}
			return nrows;
		}
	}
	return 0;
}
