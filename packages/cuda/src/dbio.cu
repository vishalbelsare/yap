#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <stdio.h>
#include "memory.h"
#include "union2.h"
#include "dbio.h"

#ifdef DATALOG
//template<class InputIterator>
//void datalogWrite(int query, InputIterator rul_str, InputIterator fin, int finalDR, int **result)
void datalogWrite(int query, vector<rulenode>::iterator rul_str, vector<rulenode>::iterator fin, int finalDR, int **result)
{
	rulenode tmprule;
	vector<rulenode>::iterator qposr;
	int *dop1, *hres;
	int cols1, res_rows, tipo;
	tmprule.name = query;
	qposr = lower_bound(rul_str, fin, tmprule, comparer);
	cols1 = qposr->num_columns;
	res_rows = cargafinal(query, cols1, &dop1);

	if(res_rows != 0)
	{	
		if(res_rows > 0)
		{
			if(finalDR)
				res_rows = unir(dop1, res_rows, cols1, &dop1, 0);
			tipo = res_rows * cols1 * sizeof(int);
			hres = (int *)malloc(tipo);
			hipMemcpy(hres, dop1, tipo, hipMemcpyDeviceToHost);
			hipFree(dop1);
			*result = hres;
		}
		else
		{
			res_rows *= -1;
			if(finalDR)
			{
				int *dop2;
				tipo = res_rows * cols1 * sizeof(int);
				reservar(&dop2, tipo); 
				hipMemcpy(dop2, dop1, tipo, hipMemcpyHostToDevice);
				free(dop1);
				res_rows = unir(dop2, res_rows, cols1, &dop2, 0);
				tipo = res_rows * cols1 * sizeof(int);
				hres = (int *)malloc(tipo);
				hipMemcpy(hres, dop2, tipo, hipMemcpyDeviceToHost);
				hipFree(dop2);
				*result = hres;
			}
			else
				*result = dop1;
		}
	}
}
#endif

#ifdef TUFFY
void postgresRead(PGconn **ret, vector<gpunode> *L, int *inpquery, char *names, int finalDR)
{
	PGresult *pgr;
	int x, y;
	int *mat, *mat2;
	char *tok, sel[1024], **qrs;
	int w, z = 0, numt, numc, numc2, start = 0, start2, val;
	PGconn *conn = PQconnectdb("host=localhost port=5432 dbname = prueba user=tuffer password=root");
	if(PQstatus(conn) != CONNECTION_OK)
    	{
        	fprintf(stderr, "Connection to database failed: %s", PQerrorMessage(conn));
		exit(1);
        }

	pgr = PQexec(conn, "Select nspname from pg_catalog.pg_namespace where oid = (select max(oid) from pg_catalog.pg_namespace)");
	sprintf(sel, "SET search_path = %s", PQgetvalue(pgr, 0, 0)); 
	PQclear(pgr);
	PQexec(conn, sel);
	tok = strtok(names, " ");	
	if(finalDR)
	{
		qrs = (char **)malloc(100 * sizeof(char *));
		while(tok != NULL)
		{
			sprintf(sel, "Select * from %s limit 0", tok);
			pgr = PQexec(conn, sel);
			numc = L->at(z).num_columns;
			if(tok[0] == 'c')
			{
				sprintf(sel, "Select ");
				numt = numc + 1;
				for(x = 1; x < numt; x++)
				{
					strcat(sel, PQfname(pgr, x));
					strcat(sel, ", ");
				}
				sel[strlen(sel)-2] = '\0';
				sprintf(sel, "%s from %s", sel, tok);
			}
			else
			{
				sprintf(sel, "Select id, Club, ");
				numt = numc + 6;
				for(x = 8; x < numt; x++)
				{
					strcat(sel, PQfname(pgr, x));
					strcat(sel, ", ");
				}
				sel[strlen(sel)-2] = '\0';
				sprintf(sel, "%s from %s", sel, tok);
			}
			PQclear(pgr);
			pgr = PQexec(conn, sel);
			numt = PQntuples(pgr);
			mat = (int *)malloc(numt * numc * sizeof(int));
			if(tok[0] == 'c')
			{
				for(x = 0; x < numt; x++)
				{
					start = x * numc;
					for(y = 0; y < numc; y++)
						mat[start + y] = atoi(PQgetvalue(pgr, x, y));
				}
			}
			else
			{
				numc2 = numc - 2;
				mat2 = (int *)malloc(numt * numc2 * sizeof(int));
				start = 0;
				start2 = 0;
				for(x = 0; x < numt; x++)
				{
					w = atoi(PQgetvalue(pgr, x, 1));
					if(w < 2)
					{
						mat[start] = atoi(PQgetvalue(pgr, x, 0));
						start++;
						mat[start] = w;
						start++;
						if(w > 0)
						{
							for(y = 2; y < numc; y++)
							{
								val = atoi(PQgetvalue(pgr, x, y));
								mat[start] = val;
								mat2[start2] = val;
								start++;
								start2++;
							}
						}
						else
						{
							for(y = 2; y < numc; y++)
							{
								val = atoi(PQgetvalue(pgr, x, y));
								mat[start] = val;
								start++;
							}
						}
					}
					else
					{
						for(y = 2; y < numc; y++)
						{
							val = atoi(PQgetvalue(pgr, x, y));
							mat2[start2] = val;
							start2++;
						}
					}
				}
				L->at(z+1).address_host_table = mat2;
				L->at(z+1).num_rows = start2 / numc2;
			}
			L->at(z).address_host_table = mat;
			L->at(z).num_rows = start / numc;
			PQclear(pgr);
			
			x = 1;
			while(inpquery[x] != -1)
			{
				if(L->at(z).name == inpquery[x])
				{
					numt = (strlen(tok) + 1) * sizeof(char);
					qrs[x] = (char *)malloc(numt);
					memcpy(qrs[x], tok, numt);
				}
				x += 2;
			}
			if(tok[0] == 'c')
			{
				tok = strtok(NULL, " ");
				z++;
			}
			else
			{
				strtok(NULL, " ");	
				tok = strtok(NULL, " ");
				z += 2;
			}
		}
	}
	else
	{
		while(tok != NULL)
		{
			sprintf(sel, "Select * from %s limit 0", tok);
			pgr = PQexec(conn, sel);
			numc = L->at(z).num_columns;
			if(tok[0] == 'c')
			{
				sprintf(sel, "Select weight, myid, ");
				start = 1;
				numt = numc + 1;
			}
			else
			{
				sprintf(sel, "Select truth, Club, atomID, ");
				start = 8;
				numt = numc + 5;
			}
			for(x = start; x < numt; x++)
			{
				strcat(sel, PQfname(pgr, x));
				strcat(sel, ", ");
			}
			sel[strlen(sel)-2] = '\0';
			sprintf(sel, "%s from %s", sel, tok);
			PQclear(pgr);
			pgr = PQexec(conn, sel);
			numt = PQntuples(pgr);
			mat = (int *)malloc(numt * numc * sizeof(int)); 
			L->at(z).weight = (double *)malloc(numt * sizeof(double));
			L->at(z).num_rows = numt;

			for(x = 0; x < numt; x++)
			{	
				start = x * numc;
				for(y = 1; y < numc; y++)
					mat[start + y] = atoi(PQgetvalue(pgr, x, y));
			}

			numt *= numc;
			double flo;
			if(tok[0] == 'c')
			{
				for(x = 0, y = 0; x < numt; x+=numc, y++)
				{
					flo = atof(PQgetvalue(pgr, y, 0));
					L->at(z).weight[y] = flo;
					if(flo > 0)
						mat[x] = y + 1;
					else
						mat[x] = -y - 1;
				}
			}
			else
			{
				for(x = 0, y = 0; x < numt; x+=numc, y++)
				{
					if(PQgetvalue(pgr, y, 0)[0] == 't')
						mat[x] = 2;
					else
						mat[x] = 1;
				}				
			}
			L->at(z).address_host_table = mat;
			numc = (strlen(tok) + 1) * sizeof(char);
			L->at(z).predname = (char *)malloc(numc);
			memcpy(L->at(z).predname, tok, numc);
			PQclear(pgr);
			tok = strtok(NULL, " ");
			z++;
		}
	}
	*ret = conn;
}

void postgresWrite(int *inpquery, int ninpf, vector<rulenode>::iterator rul_str, vector<rulenode>::iterator fin, vector<gpunode> *L, PGconn *conn, int finalDR)
{
	char sel[1024];
	double *matw = NULL;
	int qname, cols1, res_rows, tipo, *dop1;
	int x, w, z, y, *hres;
	rulenode tmprule;
	vector<rulenode>::iterator qposr;
	if(finalDR)
	{
		char file[] = "/dev/shm/mln0_atoms.csv";
		z = 0;
		int seqid = 1;
		FILE *fp;
		fp = fopen(file, "w");
		if(fp == NULL)
		{
			cerr << "Failed to create main memory temporary file, attempting to use hardrive" << endl;
			sprintf(file, "./temp/mln0_atoms.csv");
			fp = fopen(file, "w");
			if(fp == NULL)
			{
				cerr << "Failed to create main memory temporary file" << endl;
				exit(1);
			}
		}
		while((qname = inpquery[z]) != -1)
		{
			tmprule.name = qname;
			qposr = lower_bound(rul_str, fin, tmprule, comparer);
			cols1 = qposr->num_columns;
			res_rows = cargafinal(qname, cols1, &dop1);

			if(res_rows != 0)
			{
				if(res_rows < 0)
					res_rows = unir(dop1, -res_rows, cols1, &dop1, 0);  /*duplicate elimination on result*/
				else
					res_rows = unir(dop1, res_rows, cols1, &dop1, finalDR);

				tipo = res_rows * cols1 * sizeof(int);
				hres = (int *)malloc(tipo);
				hipMemcpy(hres, dop1, tipo, hipMemcpyDeviceToHost);
				hipFree(dop1);
				w = z + 1;

				strtok(qposr->rulename, "_");
				strtok(NULL, "_");
				int prid = atoi(strtok(NULL, "_"));

				for(x = 0, w = 0; x < res_rows; x++, w+=2)
				{
					if(hres[w+1])
						fprintf(fp, "%d,%d,%d,true\n", seqid, hres[w], prid);
					else
						fprintf(fp, "%d,%d,%d,false\n", seqid, hres[w], prid);
					seqid++;
				}
				free(hres);
			}
			z += 2;
		}
		fclose(fp);
		sprintf(sel, "Copy mln0_atoms(atomid,tupleID,predID,isquery) from '%s' CSV", file);
		PQexec(conn, sel);
	}
	else
	{
		while(rul_str != fin)
		{
			cols1 = rul_str->num_columns;
			res_rows = cargafinal(rul_str->name, cols1, &dop1);
			if(res_rows == 0)
			{
				rul_str++;
				continue;
			}
			res_rows = abs(res_rows);
			tipo = res_rows * cols1 * sizeof(int);
			hres = (int *)malloc(tipo);
			hipMemcpy(hres, dop1, tipo, hipMemcpyDeviceToHost);
			hipFree(dop1);

			char file[] = "/dev/shm/buffer.csv";
			FILE *fp;
			fp = fopen(file, "w");
			if(fp == NULL)
			{
				cerr << "Failed to create main memory temporary file, attempting to use hardrive" << endl;
				sprintf(file, "./temp/buffer.csv");
				fp = fopen(file, "w");
				if(fp == NULL)
				{
					cerr << "Failed to create main memory temporary file" << endl;
					exit(1);
				}
			}

			if(rul_str->rulename[0] == 'z')
			{
				char *name = rul_str->rulename + 1;
				for(x = 0; x < ninpf; x++)
				{
					if(strncmp(L->at(x).predname, name, strlen(name)) == 0)
					{
						matw = L->at(x).weight;
						break;
					}
				}

				cols1 -= 3;
				for(x = 0, z = 0; x < res_rows; x++, z+=3)
				{
					for(y = 0; y < cols1; y++, z++)
						fprintf(fp, "%d,", hres[z]);
					fprintf(fp, "%d,%lf,%d\n", hres[z], matw[abs(hres[z+1])-1], hres[z+2]);
				}
				fclose(fp);
				sprintf(sel, "Copy %s from '%s' CSV", name, file);
				PQexec(conn, sel);
			}
			else
			{
				cols1--;
				for(x = 0, z = 0; x < res_rows; x++, z++)
				{
					for(y = 0; y < cols1; y++, z++)
						fprintf(fp, "%d,", hres[z]);
					fprintf(fp, "%d\n", hres[z]);
				}
				fclose(fp);
				sprintf(sel, "Copy %s from '%s' CSV", rul_str->rulename, file);
				PQexec(conn, sel);
			}
			free(hres);
			rul_str++;
		}
	}
	PQfinish(conn);
	if(finalDR)
		clear_memory_all();
}
#endif

#ifdef ROCKIT
void mysqlRead(MYSQL **ret, int *qrs, vector<gpunode> *L, int ninpf, char *names, int finalDR)
{
	char *tok, sel[1024];
	int w, x, y, z = 0, numt, numc;
	int *mat;
	MYSQL *con = mysql_init(NULL);
	if(con == NULL)
	{
		fprintf(stderr, "mysql_init() failed\n");
      		exit(1);
	}
	mysql_options(con, MYSQL_OPT_LOCAL_INFILE, NULL);
	mysql_real_connect(con, "localhost", "root", "root", "rockit", 0, NULL, 0);
	if(finalDR)
	{
		y = 0;
		while(qrs[y] != 0)
		{
			for(z = 0; z < ninpf; z++)
			{
				if(qrs[y] == L->at(z).name)
				{
					MYSQL_ROW row;
					sprintf(sel, "Select count(*) from %s", L->at(z).predname);
					mysql_query(con, sel);
					MYSQL_RES *result = mysql_store_result(con);
					row = mysql_fetch_row(result);
					numt = atoi(row[0]);
					mysql_free_result(result);

					if(numt != L->at(z).num_rows)
					{
						liberar(L->at(z).name);
						numc = L->at(z).num_columns;
						sprintf(sel, "Select * from %s", L->at(z).predname);
						mysql_query(con, sel);
						MYSQL_RES *result = mysql_store_result(con);
						mat = (int *)malloc(numt * numc * sizeof(int));
						w = 0;
						while ((row = mysql_fetch_row(result))) 
						{
							for(x = 0; x < numc; x++, w++)
								mat[w] = atoi(row[x]);
						}

						mysql_free_result(result);
						if(L->at(z).address_host_table != NULL)
							free(L->at(z).address_host_table);
						L->at(z).address_host_table = mat;
						L->at(z).num_rows = numt;
					}
				}
			}
			y++;
		}
	}
	else
	{
		tok = strtok(names, " ");
		while(tok != NULL)
		{
			numc = L->at(z).num_columns;
			sprintf(sel, "Select * from %s", tok);
			mysql_query(con, sel);
			MYSQL_RES *result = mysql_store_result(con);
			numt = mysql_num_rows(result);

			MYSQL_ROW row;
			mat = (int *)malloc(numt * numc * sizeof(int));
			w = 0;
			if(tok[0] == 'f' && tok[1] >= '0' && tok[1] <= '9')
			{
				while ((row = mysql_fetch_row(result))) 
				{
					for(x = 1; x <= numc; x++, w++)
						mat[w] = atoi(row[x]);
				}
			}
			else
			{
				while ((row = mysql_fetch_row(result))) 
				{
					for(x = 0; x < numc; x++, w++)
						mat[w] = atoi(row[x]);
				}
			}
			mysql_free_result(result);
			L->at(z).address_host_table = mat;
			L->at(z).num_rows = numt;

			numc = (strlen(tok) + 1) * sizeof(char);
			L->at(z).predname = (char *)malloc(numc);
			strcpy(L->at(z).predname, tok);
			tok = strtok(NULL, " ");
			z++;
		}
	}
	*ret = con;
}

void mysqlWrite(vector<rulenode>::iterator rul_str, vector<rulenode>::iterator fin, vector<gpunode> *L, MYSQL *con)
{
	int x, y, z, cols1, cols2, res_rows, tipo;
	int *hres, *dop1;
	char *id, *sign, *q1, *q2;
	char sel[1024], weight[1024];
	gpunode tmpfact;
	while(rul_str != fin)
	{
		cols1 = rul_str->num_columns;
		res_rows = cargafinal(rul_str->name, cols1, &dop1);
		id = strtok(rul_str->rulename, "_");
		sprintf(sel, "create table if not exists %s(weight double, ", id);
		for(x = 0; x < cols1; x++)
		{
			sprintf(weight, "a%d char(10), ", x);
			strcat(sel, weight);
		}
		sel[strlen(sel)-2] = ')';
		strcat(sel, "ENGINE = MEMORY DEFAULT CHARSET=latin1");
		mysql_query(con, sel);
		sprintf(sel, "truncate %s", id);
		mysql_query(con, sel);

		if(res_rows == 0)
		{
			rul_str++;
			continue;
		}

		if(res_rows > 0)
		{
			tmpfact = L->at(-rul_str->referencias[rul_str->num_rows - 2] - 1);
			sign = tmpfact.predname;
			tipo = res_rows * cols1 * sizeof(int);
			hres = (int *)malloc(tipo);
			hipMemcpy(hres, dop1, tipo, hipMemcpyDeviceToHost);
			if(sign[0] == 'f' && sign[1] >= '0' && sign[1] <= '9')
				sumar(tmpfact.name, dop1, cols1, res_rows);
		}
		else
		{
			hres = dop1;
			res_rows = -res_rows;
		}

		sign = strtok(NULL, "_");
		q1 = strtok(NULL, "_");
		q2 = strtok(NULL, "_");
		if(sign[0] == '0')
			sprintf(weight, "%s.%s", q1, q2);
		else
			sprintf(weight, "-%s.%s", q1, q2);

		FILE *fp;
		char file[512];
		sprintf(file, "/dev/shm/%s.tsv", id);
		fp = fopen(file, "w");
		if(fp == NULL)
		{
			cerr << "Failed to create main memory temporary file, attempting to use hardrive" << endl;
			sprintf(file, "./temp/%s.tsv", id);
			fp = fopen(file, "w");
		}
		
		cols2 = cols1 - 1;
		for(x = 0, z = 0; x < res_rows; x++, z++)
		{
			fprintf(fp, "%s\t", weight);
			for(y = 0; y < cols2; y++, z++)
				fprintf(fp, "%d\t", hres[z]);
			fprintf(fp, "%d\n", hres[z]);
		}
		fclose(fp);

		sprintf(sel, "LOAD DATA LOCAL INFILE '%s' INTO TABLE %s", file, id);
		mysql_query(con, sel);
		rul_str++;
	}
	mysql_close(con);
}
#endif

