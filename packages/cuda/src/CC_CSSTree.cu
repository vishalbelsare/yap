#include "CC_CSSTree.h"

//return the start position of searching the key.
int CC_CSSTree::search(int key)
{
	int i=0;
	int curIndex=0;
	int curNode=0;
	int j=0;
	//search
	for(i=0;i<level;i++)
	{
		for(j=0;j<blockSize;j++)
		{
			if(ntree[curIndex+j]==-1)
				break;
			if(key<=ntree[curIndex+j])
				break;
		}
		curNode=(fanout*(curNode)+j+1);
		curIndex=curNode*blockSize;
//#ifdef DEBUG
//		cout<<curNode<<", "<<j<<", "<<ntree[curIndex]<<";   ";
//#endif
	}
	curIndex=(curNode-numNode)*blockSize;
	if(curIndex>numRecord) curIndex=numRecord-1;
	//cout<<"I: "<<curIndex<<", ";//cout<<endl;
	return curIndex;
}
