#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include "lista.h"
#include "memory.h"
extern "C" {
#include "pred.h"
}
#include "selectproyect.cu"
#include "selectproyectcpu.cpp"
#include "treeb.cu"
#include "joincpu.cpp"
#include "union2.h"
#include "unioncpu2.cpp"
#include "bpreds.h"
#include "bpredscpu.cpp"
#include "dbio.h"

#define MAXVALS 2000

#if TIMER
statinfo cuda_stats;
#endif

/*Auxiliary function to sort fact list*/
bool compare(const gpunode &r1, const gpunode &r2)
{
	return (r1.name > r2.name); 
}

/*Creates and stores the rule nodes in the rule list*/ 
void meter(vector<rulenode> *rules, predicate **p, int size)
{
	int x;
	rulenode temp;
	for(x = 0; x < size; x++)
	{
		temp.name = (*p[x]).name;
		temp.num_rows = (*p[x]).num_rows;
		temp.address_host_table = (*p[x]).address_host_table;
		temp.negatives = (*p[x]).negatives;
		temp.rulename = (*p[x]).predname;
		rules->push_back(temp);
	}
}

/*Find the first predicate that has a certain variable val*/
int closestpred(int *arr, int size, int val)
{
	int x;
	for(x = 1; x < size; x++)
	{
		if(arr[x] > val)
			break;
	}
	return x - 2;
}

/*template<class InputIterator>
void checkjoins(InputIterator rules, InputIterator end)
{
			x = 1;
			while(rules->rule_name[x] < 0)
				x++;
			pos = rules->rule_names[x] + 1;

			for(y = x + 1; y < total; y++)
			{
				bs = rules->address_host_table[pos];
				pos2 = rules->rule_names[x+1] + 1;
				while((bs2 = rules->address_host_table[pos2]) > 0)
				{
					if(bs == bs2)
					{
						flag = 1;
						break;
					}
					pos2++;
				}
			}
}*/

/*Move comparison predicates based on their variables. They are moved 
after the first normal predicate that has one (in case of variable vs constant comparison)
or both (variable vs variable) variables found in the comparison. */
template<class InputIterator>
void movebpreds(InputIterator rules, InputIterator end)
{
	int x, y, subs, total, pos, fin;
	int cont, cont2, cont3;
	int p1 = -1, p2 = -1;
	int move[MAXVALS], move2[MAXVALS], rest[MAXVALS];
	
	while(rules != end)
	{
		total = rules->num_rows - 1;
		cont = total * sizeof(int2);
		rules->preds = (int **)malloc(total * sizeof(int *));
		rules->numpreds = (int2 *)malloc(cont);
		memset(rules->numpreds, 0x0, cont);
	
		if(rules->totalpreds > 0)
		{
			cont2 = MAXVALS * sizeof(int);
			for(x = 0; x < total; x++)
				rules->preds[x] = (int *)malloc(cont2);

			total = rules->num_rows + rules->totalpreds;
			cont = 0;
			cont2 = 0;
			cont3 = 0;
			for(x = 0; x < total; x++)
			{
				subs = rules->rule_names[x+1] - rules->rule_names[x];
				if(rules->address_host_table[rules->rule_names[x]] > 0)
				{
					memcpy(rest + cont, rules->address_host_table + rules->rule_names[x], subs * sizeof(int));
					cont += subs;
				}
				else
				{
					pos = rules->rule_names[x] + 1;
					subs--;
					if(rules->address_host_table[pos] > 0 && rules->address_host_table[pos+1] > 0)
					{
						memcpy(move2 + cont3, rules->address_host_table + rules->rule_names[x], subs * sizeof(int));
						cont3 += subs;
					}
					else
					{
						memcpy(move + cont2, rules->address_host_table + rules->rule_names[x], subs * sizeof(int));
						cont2 += subs;
					}
				}
			}
			memcpy(rules->address_host_table, rest, cont * sizeof(int));

			pos = 1;
			for(x = 1; x <= total; x++)
			{
				while(rest[pos] != 0)
					pos++;
				pos++;
				rules->rule_names[x] = pos;
			}

			pos = 1;
			total = 3 * sizeof(int);
			fin = cont2 / 3;
			cont2 = 0;
			for(x = 0; x < fin; x++)
			{
				subs = move[pos];
				if(subs > 0)
				{
					for(y = rules->rule_names[1] + 1; y < rules->rule_names[rules->num_rows]; y++)
					{
						if(rules->address_host_table[y] == subs)
						{
							p1 = y;
							break;
						}
					}
				}
				else
				{
					pos++;
					subs = move[pos];
					for(y = rules->rule_names[1] + 1; y < rules->rule_names[rules->num_rows]; y++)
					{
						if(rules->address_host_table[y] == subs)
						{
							p1 = y;
							break;
						}
					}
				}
				pos += 2;
				cont = closestpred(rules->rule_names, rules->num_rows, p1);

				memcpy(rules->preds[cont] + rules->numpreds[cont].x, move + cont2, total);
				rules->numpreds[cont].x += 3;
				cont2 += 3;
			}

			pos = 1;
			fin = cont3 / 3;
			cont3 = 0;
			for(x = 0; x < fin; x++)
			{
				subs = move2[pos];
				for(y = rules->rule_names[1] + 1; y < rules->rule_names[rules->num_rows]; y++)
				{
					if(rules->address_host_table[y] == subs)
					{
						p1 = y;
						break;
					}
				}
				pos++;
				subs = move2[pos];
				for(y = rules->rule_names[1] + 1; y < rules->rule_names[rules->num_rows]; y++)
				{
					if(rules->address_host_table[y] == subs)
					{
						p2 = y;
						break;
					}
				}
				pos += 2;
				cont = closestpred(rules->rule_names, rules->num_rows, p1);
				cont2 = closestpred(rules->rule_names, rules->num_rows, p2);
				if(cont == cont2)
				{
					memcpy(rules->preds[cont] + rules->numpreds[cont].x, move2 + cont3, total);
					rules->numpreds[cont].x += 3;
				}
				else
				{
					if(cont > cont2)
					{
						memcpy(rules->preds[cont] + rules->numpreds[cont].x + rules->numpreds[cont].y, move2 + cont3, total);
						rules->numpreds[cont].y += 3;
					}
					else
					{
						memcpy(rules->preds[cont2] + rules->numpreds[cont2].x + rules->numpreds[cont2].y, move2 + cont3, total);
						rules->numpreds[cont2].y += 3;
					}
				}
				cont3 += 3;
			}
		}
		rules++;
	}
}

/*Mark the location of predicate names in each rule*/
template<class InputIterator>
void nombres(InputIterator rules, InputIterator end)
{
	int x, pos;
	while(rules != end)
	{
		pos = 1;
		rules->rule_names = (int *)malloc(sizeof(int) * (rules->num_rows + 1));
		rules->rule_names[0] = 0;
		rules->totalpreds = 0;
		for(x = 1; x <= rules->num_rows; x++)
		{
			while(rules->address_host_table[pos] != 0)
				pos++;
			pos++;
			rules->rule_names[x] = pos;
			if(x < rules->num_rows && rules->address_host_table[pos] < 0)
				rules->totalpreds++;
		}
		rules->num_columns = rules->rule_names[1] - 2;
		rules->num_rows -= rules->totalpreds; /*quita los builtin del total*/
		rules++;
	}
}

/*Determine if a predicate refers to a fact or a rule*/
template<class InputIterator, class RulesIterator>
void referencias(InputIterator facts, InputIterator fend, RulesIterator rules, RulesIterator end)
{
	vector<gpunode>::iterator res1;
	gpunode n1;
	vector<rulenode>::iterator actual = rules, res2;
	rulenode n2;
	int x, temp, cont;
	while(actual != end)
	{
		actual->referencias = (int *)malloc((actual->num_rows - 1) * sizeof(int));
		cont = 0;
		for(x = 1; x < actual->num_rows; x++)
		{
			temp = actual->address_host_table[actual->rule_names[x]];
			if(temp == actual->name)
			{
				n1.name = temp;
				res1 = lower_bound(facts, fend, n1, compare);
				if (res1 != fend && res1->name == temp)
				{			
					actual->referencias[x-1] = -(res1 - facts) - 1;
					cont++;
				}
				else
					actual->referencias[x-1] = actual - rules;
			}			
			else
			{
				n2.name = temp;
				res2 = lower_bound(rules, end, n2, comparer);
				if(res2 != end && res2->name == temp)
					actual->referencias[x-1] = res2 - rules;
				else
				{
					n1.name = temp;
					res1 = lower_bound(facts, fend, n1, compare);
					actual->referencias[x-1] = -(res1 - facts) - 1;
					cont++;
				}
			}
		}
		if(cont == (x - 1))
			actual->gen_ant = 0;
		else
			actual->gen_ant = -1;
		actual->gen_act = 0;
		actual++;
	}
}

/*Creates an array for each selections found in a predicate for all rules*/
template<class InputIterator>
void seleccion(InputIterator actual, InputIterator end)
{
	int x, y, ini, temp, sl, tam, cont;
	int pv[MAXVALS];
	while(actual != end)
	{
		temp = actual->num_rows - 1;
		actual->select = (int **)malloc(temp * sizeof(int *));
		actual->numsel = (int *)malloc(temp * sizeof(int));
		for(x = 1; x < actual->num_rows; x++)
		{
			ini = actual->rule_names[x];
			if(actual->address_host_table[ini] < 0)
				continue;
			ini++;
			cont = 0;
			for(y = ini; y < (actual->rule_names[x+1] - 1); y++)
			{
				temp = actual->address_host_table[y];
				if(temp < 0)
				{
					pv[cont] = y - ini;
					cont++;
					pv[cont] = -temp;
					cont++;
				}
			}
			sl = x - 1;
			tam = cont * sizeof(int);
			actual->select[sl] = (int *)malloc(tam);
			memcpy(actual->select[sl], pv, tam);
			actual->numsel[sl] = cont;
		}
		actual++;
	}
}

int notin(int bus, int *array, int size)
{
	int x;
	for(x = 0; x < size; x++)
	{
		if(array[x] == bus)
			return 0;
	}
	return 1;
}

int2 columnsproject(int *first, int tam, int *rule, int ini, int fin, int sini, int sfin, int **res, int **newrule, int **preds, int2 *nump, int pact, int ptot)
{
	int x, y, z, temp, size;
	int pv[MAXVALS], pv2[MAXVALS];
	int2 ret = make_int2(0, 0);
	for(x = 0; x < tam; x++)
	{
		y = 0;
		temp = first[x];
		for(y = 0; y < ini; y++)
		{
			if(temp == rule[y] && temp > 0) /*added condition to avoid constants*/
			{
				if(notin(temp, pv, ret.y))
				{
					pv[ret.y] = temp;
					pv2[ret.y] = x;
					ret.y++;
				}
				break;
			}
		}
		if(y != ini)
			continue;
		for(y = sfin + 1; y < fin; y++)
		{
			if(temp == rule[y] && temp > 0)
			{
				if(notin(temp, pv, ret.y))
				{
					pv[ret.y] = temp;
					pv2[ret.y] = x;
					ret.y++;
				}
				break;
			}
		}
		if(y != fin)
			continue;
		for(y = pact; y < ptot; y++)
		{
			size = nump[y].x + nump[y].y;
			for(z = 1; z < size; z+=3)
			{
				if((temp == preds[y][z] || temp == preds[y][z+1]) && temp > 0)
				{
					if(notin(temp, pv, ret.y))
					{
						pv[ret.y] = temp;
						pv2[ret.y] = x;
						ret.y++;
					}
					break;
				}
			}
			if(z != size)
				break;
		}

	}
	ret.x = ret.y;
	for(x = sini; x < sfin; x++)
	{
		y = 0;
		temp = rule[x];
		for(y = 0; y < ini; y++)
		{
			if(temp == rule[y] && temp > 0)
			{
				if(notin(temp, pv, ret.y))
				{
					pv[ret.y] = temp;
					pv2[ret.y] = x - sini;
					ret.y++;
				}
				break;
			}
		}
		if(y != ini)
			continue;
		for(y = sfin + 1; y < fin; y++)
		{
			if(temp == rule[y] && temp > 0)
			{
				if(notin(temp, pv, ret.y))
				{
					pv[ret.y] = temp;
					pv2[ret.y] = x - sini;
					ret.y++;
				}
				break;
			}
		}
		for(y = pact; y < ptot; y++)
		{
			size = nump[y].x + nump[y].y;
			for(z = 1; z < size; z+=3)
			{
				if((temp == preds[y][z] || temp == preds[y][z+1]) && temp > 0)
				{
					if(notin(temp, pv, ret.y))
					{
						pv[ret.y] = temp;
						pv2[ret.y] = x - sini;
						ret.y++;
					}
					break;
				}
			}
			if(z != size)
				break;
		}
	}

	temp = ret.y * sizeof(int);
	free(*newrule);
	*newrule = (int *)malloc(temp);
	memcpy(*newrule, pv, temp);
	*res = (int *)malloc(temp);
	memcpy(*res, pv2, temp);	
	return ret;
}

/*Determines all possible joins between two adjacent predicates*/
int wherejoin(int *tmprule, int tmplen, int *rule, int tam2, int **res)
{
	int x, y, temp;
	int cont = 0;
	int joins[MAXVALS];
	for(x = 0; x < tmplen; x++)
	{
		for(y = 0; y < tam2; y++)
		{
			if(rule[y] > 0 && tmprule[x] == rule[y])
			{
				joins[cont] = x;
				cont++;
				joins[cont] = y;
				cont++;
				break;
			}
		}
	}

	temp = cont * sizeof(int);
	*res = (int *)malloc(temp);
	memcpy(*res, joins, temp);
	return cont;
}

template<class InputIterator>
void builtinmark(InputIterator actual, InputIterator end)
{
	int x, y, z;
	int ini, fin, bus;

	while(actual != end)
	{
		for(x = 0; x < (actual->num_rows - 1); x++)
		{
			ini = actual->rule_names[x+1] + 1;
			fin = actual->rule_names[x+2] - 1;

			for(y = 1; y < actual->numpreds[x].x; y += 2)
			{
				bus = actual->preds[x][y];
				if(bus > 0)
				{
					for(z = ini; z < fin; z++)
					{
						if(actual->address_host_table[z] == bus)
						{
							actual->preds[x][y] = z - ini;
							break;
						}
					}
				}
				y++;
				bus = actual->preds[x][y];
				if(bus > 0)
				{
					for(z = ini; z < fin; z++)
					{
						if(actual->address_host_table[z] == bus)
						{
							actual->preds[x][y] = z - ini;
							break;
						}
					}
				}
			}
		}
		actual++;
	}
}

/*Creates arrays for each predicate with the positions of join and projection columns*/
template<class InputIterator>
void proyeccion(InputIterator actual, InputIterator end)
{
	int x, y, ini, fin, total, numjoins, temp, rulestart, ruleend, malptr;
	int fjoin[MAXVALS];
	int *pv, *res;
	int2 pos;
	while(actual != end)
	{
		numjoins = actual->num_rows - 2;
		if(numjoins < 1)
		{
			actual->projpos = (int2 *)malloc(sizeof(int2));
			ini = actual->rule_names[1] + 1;
			fin = actual->rule_names[2] - 1;

			if(actual->numsel[0] == 0 && actual->numselfj[0] == 0 && actual->num_columns == (fin - ini) && actual->numpreds[0].x == 0)
			{
				for(x = 1, y = actual->num_columns + 3; x <= actual->num_columns; x++, y++)
				{
					if(actual->address_host_table[x] != actual->address_host_table[y])
						break;
				}
				if(x > actual->num_columns)
				{
					pos.x = -1;
					pos.y = -1;
					actual->projpos[0] = pos;
					actual++;
					continue;
				}
			}

			actual->project = (int **)malloc(sizeof(int *));
			pos.x = 0;
			for(x = 1; x <= actual->num_columns; x++)
			{
				temp = actual->address_host_table[x];
				for(y = ini; y < fin; y++)
				{
					if(temp == actual->address_host_table[y])
					{
						fjoin[pos.x] = y - ini;
						pos.x++;
						break;
					}
				}
			}			

			temp = pos.x * sizeof(int);
			actual->project[0] = (int *)malloc(temp);
			memcpy(actual->project[0], fjoin, temp);
			pos.y = pos.x;
			actual->projpos[0] = pos;
			actual++;
			continue;
		}
		malptr = numjoins * sizeof(int *);
		actual->project = (int **)malloc(malptr);
		actual->projpos = (int2 *)malloc(numjoins * sizeof(int2));
		actual->wherejoin = (int **)malloc(malptr);
		actual->numjoin = (int *)malloc(numjoins * sizeof(int));
		ini = actual->rule_names[1] + 1;
		total = actual->num_rows + actual->totalpreds;
		fin = actual->rule_names[total] - 1;
		pos.y = actual->rule_names[2] - actual->rule_names[1] - 2;
		temp = pos.y * sizeof(int);
		pv = (int *)malloc(temp);
		memcpy(pv, actual->address_host_table + actual->rule_names[1] + 1, temp);

		for(x = 2, y = 0; x <= numjoins; x++, y++)
		{
			rulestart = actual->rule_names[x] + 1;
			ruleend = actual->rule_names[x+1] - 1;
			temp = wherejoin(pv, pos.y, actual->address_host_table + rulestart, ruleend - rulestart, &res);
			actual->wherejoin[y] = res;
			actual->numjoin[y] = temp;
			pos = columnsproject(pv, pos.y, actual->address_host_table, ini, fin, rulestart, ruleend, &res, &pv, actual->preds, actual->numpreds, y + 2, actual->num_rows);
			actual->project[y] = res;
			actual->projpos[y] = pos;
		}

		rulestart = actual->rule_names[actual->num_rows-1] + 1;
		ruleend = actual->rule_names[actual->num_rows] - 1; 
		temp = wherejoin(pv, pos.y, actual->address_host_table + rulestart, ruleend - rulestart, &res);
		actual->wherejoin[y] = res;
		actual->numjoin[y] = temp;
		numjoins--;

		pos.x = 0;
		for(x = 1; x <= actual->num_columns; x++)
		{
			temp = actual->address_host_table[x];
			for(y = 0; y < pos.y; y++)
			{
				if(temp == pv[y])
				{
					fjoin[pos.x] = y + 1;
					pos.x++;
					break;
				}
			}
			if(y != pos.y)
				continue;
			for(y = rulestart; y < ruleend; y++)
			{
				if(temp == actual->address_host_table[y])
				{
					fjoin[pos.x] = -(y - rulestart + 1);
					pos.x++;
					break;
				}
			}
		}
		temp = pos.x * sizeof(int);	
		actual->project[numjoins] = (int *)malloc(temp);
		memcpy(actual->project[numjoins], fjoin, temp);
		pos.y = pos.x;
		actual->projpos[numjoins] = pos;
		actual++;
	}
}

/*Creates an array for each predicate for all rules where selfjoin positions are stored*/
template<class InputIterator>
void selfjoin(InputIterator actual, InputIterator end)
{
	int x, y, z;
	int cont, tam, temp, size, pos;
	int fjoin[MAXVALS], rulecpy[MAXVALS];
	while(actual != end)
	{
		size = actual->num_rows - 1;
		actual->selfjoin = (int **)malloc(size * sizeof(int *));
		actual->numselfj = (int *)malloc(size * sizeof(int));
		for(x = 1; x <= size; x++)
		{
			pos = actual->rule_names[x];
			if(actual->address_host_table[pos] < 0)
				continue;
			tam = actual->rule_names[x+1] - actual->rule_names[x] - 2;
			memcpy(rulecpy, actual->address_host_table + pos + 1, tam * sizeof(int));
			cont = 0;
			for(y = 0; y < tam; y++)
			{
				temp = rulecpy[y];
				if(temp > -1)
				{
					for(z = y + 1; z < tam; z++)
					{
						if(temp == rulecpy[z])
						{
							fjoin[cont] = y;
							cont++;
							fjoin[cont] = z;
							cont++;
							rulecpy[z] = -1;
							for(z++; z < tam; z++)
							{
								if(temp == rulecpy[z])
								{
									fjoin[cont] = z;
									cont++;
									rulecpy[z] = -1;
								}
							}
							fjoin[cont] = -1;
							cont++;
						}
					}
				}
			}
			temp = x - 1;
			tam = cont * sizeof(int);
			actual->selfjoin[temp] = (int *)malloc(tam);
			memcpy(actual->selfjoin[temp], fjoin, tam);
			actual->numselfj[temp] = cont;
		}
		actual++;
	}
}

void consulta(int *query, int qsize, int qname, rulenode *res)
{
	int sel[MAXVALS], pro[MAXVALS];
	int temp, cont1 = 0, cont2 = 0, size;
	int x, y;
	res->numsel = (int *)malloc(sizeof(int));
	res->numselfj = (int *)malloc(sizeof(int));
	for(x = 0; x < qsize; x++)
	{
		temp = query[x];
		if(temp < 0)
		{
			sel[cont1] = x;
			cont1++;
			sel[cont1] = -temp;
			cont1++;
		}
		else
		{
			pro[cont2] = x;
			cont2++;
		}
	}
	res->numsel[0] = cont1;
	res->num_columns = cont2;
	if(cont1 > 0)
	{
		size = cont1 * sizeof(int);
		res->select = (int **)malloc(sizeof(int *));
		res->select[0] = (int *)malloc(size);
		memcpy(res->select[0], sel, size);
		cont1 = 0;
	}
	if(cont2 > 0)
	{
		size = cont2 * sizeof(int);
		res->project = (int **)malloc(sizeof(int *));
		res->project[0] = (int *)malloc(size);
		memcpy(res->project[0], pro, size);
	}
	for(x = 0; x < qsize; x++)
	{
		temp = query[x];
		if(temp > -1)
		{
			for(y = x + 1; y < qsize; y++)
			{
				if(temp == query[y])
				{
					sel[cont1] = x;
					cont1++;
					sel[cont1] = y;
					cont1++;
					query[y] = -1;
					for(y++; y < qsize; y++)
					{
						if(temp == query[y])
						{
							sel[cont1] = y;
							cont1++;
							query[y] = -1;
						}
					}
					sel[cont1] = -1;
					cont1++;
				}
			}
		}
	}
	res->numselfj[0] = cont1;
	if(cont1 > 0)
	{
		size = cont1 * sizeof(int);
		res->selfjoin = (int **)malloc(sizeof(int *));
		res->selfjoin[0] = (int *)malloc(size);
		memcpy(res->selfjoin[0], sel, size);
	}
}

/*Discards finished rules based on their predicates. Rules with only facts finish after one iteration. Rules with other rules as predicates finish when at least one of the rules finishes.*/
void discardRules(list<rulenode> *reglas, int itr)
{
	list<rulenode>::iterator begin = reglas->begin();
	list<rulenode>::iterator end = reglas->end();
	list<rulenode>::iterator busqueda, rul_act = begin;
	int x, num_refs, tipo;
	rulenode completed;
	while(rul_act != end)
	{
		if(rul_act->gen_act == -1 || rul_act->gen_ant == -1)
		{
			rul_act++;
			continue;
		}
		if(rul_act->gen_act == 0)
		{
			rul_act->gen_act = -1;
			rul_act++;
			continue;
		}
		num_refs = rul_act->num_rows - 1;
		for(x = 0; x < num_refs; x++)
		{
			tipo = rul_act->referencias[x];
			if(tipo < 0)
				continue;
			completed.name = rul_act->address_host_table[rul_act->rule_names[x+1]];
			if(!binary_search(begin, end, completed, comparer))
			{
				rul_act->gen_act = -1;
				break;
			}

			tipo = rul_act->name;
			if(generadas(tipo, rul_act->num_rows, rul_act->num_columns, itr))
			{
				rul_act->gen_act = -1;
				busqueda = rul_act;
				busqueda++;
				while(busqueda != end && busqueda->name == tipo)
				{
					busqueda->gen_act = -1;
					busqueda++;
				}
			}
			break;
		}
		if(x == num_refs)
			rul_act->gen_act = -1;
		rul_act++;
	}
	rul_act = begin;
	while(rul_act != end)
	{
		if(rul_act->gen_act == -1)
			rul_act = reglas->erase(rul_act);
		else
			rul_act++;
	}
}

/*Display information about each rule*/
template<class InputIterator>
void mostrarcontenido(InputIterator actual, InputIterator end)
{
	int y, z, num;
	cout << "AUX INICIO" << endl;
	cout << "tamanio = " << end - actual << endl;
	while(actual != end)
	{
		cout << "name = " << actual->name << endl;
		cout << "rule_names = ";
		for(y = 0; y <= actual->num_rows; y++)
			cout << actual->rule_names[y] << " ";
		cout << endl << "referencias = ";
		num = actual->num_rows - 1;
		for(y = 0; y < num; y++)
			cout << actual->referencias[y] << " ";
		cout << endl << "select = " << endl;
		for(y = 0; y < num; y++)
		{	
			cout << actual->numsel[y] << ": ";
			for(z = 0; z < actual->numsel[y]; z++)
				cout << actual->select[y][z] << " ";
			cout << endl;
		}
		cout << "selfjoins = " << endl;
		for(y = 0; y < num; y++)
		{	
			cout << actual->numselfj[y] << ": ";
			for(z = 0; z < actual->numselfj[y]; z++)
				cout << actual->selfjoin[y][z] << " ";
			cout << endl;
		}
		num--;
		cout << "project = " << endl;
		for(y = 0; y < num; y++)
		{	
			cout << actual->projpos[y].x << " " << actual->projpos[y].y << ": ";
			for(z = 0; z < actual->projpos[y].y; z++)
				cout << actual->project[y][z] << " ";
			cout << endl;
		}
		cout << "wherejoin = " << endl;
		for(y = 0; y < num; y++)
		{	
			cout << actual->numjoin[y] << ": ";
			for(z = 0; z < actual->numjoin[y]; z++)
				cout << actual->wherejoin[y][z] << " ";
			cout << endl;
		}
		actual++;
	}	
	cout << "AUX FIN" << endl;
}

extern "C"
 void Cuda_Statistics(void)
{
  cerr << "GPU Statistics" << endl;
#if TIMER
  cerr << "Called " << cuda_stats.calls << "times." << endl;
  cerr << "GPU time " << cuda_stats.total_time << "msec." << endl;
  cerr << "Longest call " << cuda_stats.max_time << "msec." << endl;
  cerr << "Fastest call " << cuda_stats.min_time << "msec." << endl << endl;
  cerr << "Steps" << endl;
  cerr << "    Select First: " << cuda_stats.select1_time << " msec." << endl;
  cerr << "    Select Second: " << cuda_stats.select2_time << " msec." << endl;
  cerr << "    Sort: " << cuda_stats.sort_time << " msec." << endl;
  cerr << "    Join: " << cuda_stats.join_time << " msec." << endl;
  cerr << "    Union: " << cuda_stats.union_time << " msec." << endl;
  cerr << "    Built-in: " << cuda_stats.pred_time << " msec." << endl << endl;
  cerr << "Operations" << endl;
  cerr << "    Joins: " << cuda_stats.joins << "." << endl;
  cerr << "    Selects/Projects: " << cuda_stats.selects << "." << endl;
  cerr << "    Unions: " << cuda_stats.unions << "." << endl;
  cerr << "    Built-ins: " << cuda_stats.builtins << "." << endl << endl;
#endif
}

vector<gpunode> L;

extern "C"
int Cuda_Eval(predicate **inpfacts, int ninpf, predicate **inprules, int ninpr, int *inpquery, int **result, char *names, int finalDR)
{
	hipSetDevice(0);
	vector<rulenode> rules;
	int x;

#if TIMER
	cuda_stats.calls++;
#endif

	#ifdef ROCKIT
	if(finalDR)
		ninpf *= -1;
	else //this else makes the 'for' conditional
	#else
	L.clear();
	#endif
	for(x = 0; x < ninpf; x++)
		L.push_back(*inpfacts[x]);
	meter(&rules, inprules, ninpr);

	#ifdef TUFFY
	PGconn *conn = NULL;
	postgresRead(&conn, &L, inpquery, names, finalDR);
	#endif
	#ifdef ROCKIT
	MYSQL *con = NULL;
	mysqlRead(&con, inpquery, &L, ninpf, names, finalDR);
	#endif

	int res_rows = 0, rows1, rows2;
	int tipo;
	int *dop1, *dop2, *res;
	
	vector<rulenode>::iterator rul_str, fin;
	sort(L.begin(), L.end(), compare);
	sort(rules.begin(), rules.end(), comparer);
	rul_str = rules.begin();
	fin = rules.end();

	nombres(rul_str, fin); /*preprocessing*/
	movebpreds(rul_str, fin);
	referencias(L.begin(), L.end(), rul_str, fin);
	seleccion(rul_str, fin);
	selfjoin(rul_str, fin);
	proyeccion(rul_str, fin);
	builtinmark(rul_str, fin);
	//mostrarcontenido(rul_str, fin);

	list<rulenode>::iterator rul_act, busqueda;
	list<rulenode> reglas(rul_str, fin);
	
	//cargareglas(&rules, qname, &reglas);
	//mostrareglas(reglas);

	gpunode tmpfact;
	rulenode tmprule;
	int name1, filas1, cols1, isfact1, name2, filas2, cols2, isfact2;
	int *table1, *table2;
	int num_refs, itr = 0, genflag = 0;
	vector<gpunode>::iterator qposf;
	vector<rulenode>::iterator qposr;

#if TIMER
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif

	while(reglas.size()) /*Here's the main loop*/
	{
		rul_act = reglas.begin();

		while(rul_act != reglas.end()) /*Here's the loop that evaluates each rule*/
		{
			tipo = rul_act->referencias[0];

			//cout << "tipo top = " << tipo << endl;

			if(tipo < 0)
			{
				tmpfact = L.at(-tipo - 1);
				name1 = tmpfact.name;
				filas1 = tmpfact.num_rows;
				cols1 = tmpfact.num_columns;
				isfact1 = 1;
				table1 = tmpfact.address_host_table;
			}
			else
			{
				tmprule = rules.at(tipo);
				name1 = tmprule.name;
				filas1 = tmprule.num_rows;
				cols1 = tmprule.num_columns;
				isfact1 = 0;
				table1 = NULL;
			}

			//cout << "inicio " << rul_act->name << endl << "name1 = " << name1 << " filas1 = " << filas1 << " cols1 = " << cols1 << " isfact1 = " << isfact1 << endl;

			rows1 = cargar(name1, filas1, cols1, isfact1, table1, &dop1, itr);

			//cout << "rows1 = " << rows1 << endl;

			if(rows1 == 0)
			{
				rul_act->gen_act = 0;
				rul_act++;
				continue;
			}
			
			if(rul_act->num_rows < 3)
			{	
				if(rul_act->projpos[0].x == -1)
				{
					num_refs = rows1 * cols1 * sizeof(int);
					reservar(&res, num_refs);
					hipMemcpyAsync(res, dop1, num_refs, hipMemcpyDeviceToDevice);
					registrar(rul_act->name, cols1, res, rows1, itr, 1);
					genflag = 1;
					rul_act->gen_ant = rul_act->gen_act;
					rul_act->gen_act = rows1;

					if(isfact1)
					{
						tmprule.name = name1;
						qposr = lower_bound(rul_str, fin, tmprule, comparer);
						if (qposr != fin && qposr->name == name1)
							rul_act->referencias[0] =  qposr - rul_str;
					}
				}
				else
				{
					res_rows = selectproyect(dop1, rows1, cols1, rul_act->num_columns, rul_act->select[0], rul_act->numsel[0], rul_act->selfjoin[0], rul_act->numselfj[0], rul_act->preds[0], rul_act->numpreds[0].x, rul_act->project[0], &res, finalDR);

					//cout << "name = " << rul_act->name << " res_rows = " << res_rows << endl; 

					if(res_rows > 0)
					{
						registrar(rul_act->name, rul_act->num_columns, res, res_rows, itr, 1);
						genflag = 1;
						rul_act->gen_ant = rul_act->gen_act;
						rul_act->gen_act = res_rows;
						
						if(isfact1)
						{
							tmprule.name = name1;
							qposr = lower_bound(rul_str, fin, tmprule, comparer);
							if (qposr != fin && qposr->name == name1)
								rul_act->referencias[0] =  qposr - rul_str;
						}
					}
					else
						rul_act->gen_act = 0;
				}
				rul_act++;
				continue;
			}

			tipo = rul_act->referencias[1];
			if(tipo < 0)
			{
				tmpfact = L.at(-tipo - 1);
				name2 = tmpfact.name;
				filas2 = tmpfact.num_rows;
				cols2 = tmpfact.num_columns;
				isfact2 = 1;
				table2 = tmpfact.address_host_table;
			}
			else
			{
				tmprule = rules.at(tipo);
				name2 = tmprule.name;
				filas2 = tmprule.num_rows;
				cols2 = tmprule.num_columns;
				isfact2 = 0;
				table2 = NULL;
			}

			//cout << "name2 = " << name2 << " filas2 = " << filas2 << " cols2 = " << cols2 << " isfact2 = " << isfact2 << " itr = " << itr << endl;

			rows2 = cargar(name2, filas2, cols2, isfact2, table2, &dop2, itr);
			
			//cout << "rows2 = " << rows2 << endl;

			res = NULL;
			if(rows2 == 0)
			{
				if(rul_act->negatives[2])
				{
					if(rul_act->num_rows == 3)
						project(dop1, rows1, cols1, rul_act->num_columns, rul_act->project[0], &res, 1);
					else
						project(dop1, rows1, cols1, rul_act->projpos[0].x, rul_act->project[0], &res, 0);
					res_rows = rows1;
				}
				else
				{
					rul_act->gen_act = 0;
					rul_act++;
					continue;
				}
			}
			else
			{
				#ifdef ROCKIT
				if(rul_act->numjoin[0] == 0)
				{
					juntar(dop1, dop2, rows1, rows2, cols1, cols2, rul_act->project[0], rul_act->num_columns, &res);
					registrar(rul_act->name, rul_act->num_columns, res, rows1 * rows2, itr, 1);
					rul_act++;
					continue;
				}
				#endif
				res_rows = join(dop1, dop2, rows1, rows2, cols1, cols2, rul_act, 0, 1, &res, finalDR);
			}

			//cout << "res_rows = " << res_rows << " num_refs = " << rul_act->num_rows << " numcols = " << rul_act->projpos[0].y << endl;

			if(res_rows == 0)
			{
				rul_act->gen_act = 0;
				rul_act++;
				continue;
			}

			num_refs = rul_act->num_rows - 1;
			for(x = 2; x < num_refs; x++)
			{
				tipo = rul_act->referencias[x];
				if(tipo < 0)
				{
					tmpfact = L.at(-tipo - 1);
					name2 = tmpfact.name;
					filas2 = tmpfact.num_rows;
					cols2 = tmpfact.num_columns;
					isfact2 = 1;
					table2 = tmpfact.address_host_table;
				}
				else
				{
					tmprule = rules.at(tipo);
					name2 = tmprule.name;
					filas2 = tmprule.num_rows;
					cols2 = tmprule.num_columns;
					isfact2 = 0;
					table2 = NULL;
				}

				//cout << "name2 = " << name2 << " filas2 = " << filas2 << " cols2 = " << cols2 << " isfact2 = " << isfact2 << " itr = " << itr << endl;

				rows2 = cargar(name2, filas2, cols2, isfact2, table2, &dop2, itr);

				//cout << "rows = " << x << " " << rows2 << endl;
				
				if(rows2 == 0)
				{
					if(rul_act->negatives[x+1])
					{
						if(x == rul_act->num_rows - 2)
							project(res, res_rows, rul_act->projpos[x-2].y, rul_act->num_columns, rul_act->project[x-1], &res, 1);
						else
							project(res, res_rows, rul_act->projpos[x-2].y, rul_act->projpos[x-1].x, rul_act->project[x-1], &res, 0);
						continue;
					}
					break;
				}

				res_rows = join(res, dop2, res_rows, rows2, rul_act->projpos[x-2].y, cols2, rul_act, x-1, 0, &res, finalDR);

				//cout << "resrows = " << res_rows << endl;
				
			}
#ifdef ROCKIT
			if(x == num_refs)
				registrar(rul_act->name, rul_act->num_columns, res, res_rows, itr, 1);
			rul_act++;
		}
		reglas.clear();
	}
#else
			if(x == num_refs)
			{
				#ifdef TIMER
				hipEvent_t start2, stop2;
				hipEventCreate(&start2);
				hipEventCreate(&stop2);
				hipEventRecord(start2, 0);
				#endif

				//cout << rul_act->name << " res_rows = " << res_rows << endl;

				if(finalDR)
					res_rows = unir(res, res_rows, rul_act->num_columns, &res, 0);

				#ifdef TIMER
				hipEventRecord(stop2, 0);
				hipEventSynchronize(stop2);
				hipEventElapsedTime(&time, start2, stop2);
				hipEventDestroy(start2);
				hipEventDestroy(stop2);
				//cout << "Union = " << time << endl;
				cuda_stats.union_time += time;
				#endif					
	
				//cout << "despues de unir = " << res_rows << endl;

				registrar(rul_act->name, rul_act->num_columns, res, res_rows, itr, 1);
				genflag = 1;
				rul_act->gen_ant = rul_act->gen_act;
				rul_act->gen_act = res_rows;

				for(x = 0; x < num_refs; x++)
				{
					if(rul_act->referencias[x] < 0)
					{
						tmprule.name = rul_act->address_host_table[rul_act->rule_names[x+1]];
						qposr = lower_bound(rul_str, fin, tmprule, comparer);
						if (qposr != fin && qposr->name == tmprule.name)
							rul_act->referencias[x] =  qposr - rul_str;
					}
				}
			}
			else
				rul_act->gen_act = 0;
			rul_act++;
		}
		if(genflag != 1)
			break;
		else
			genflag = 0;
		discardRules(&reglas, itr);

		//cout << "itr " << itr << endl;

		itr++;
	}
#endif

	#ifdef DATALOG
	datalogWrite(inpquery[0], rul_str, fin, finalDR, result);
	#else
	res_rows = 0;
	#endif
	#ifdef TUFFY
	postgresWrite(inpquery, ninpf, rul_str, fin, &L, conn, finalDR);
	#endif
	#ifdef ROCKIT
	mysqlWrite(rul_str, fin, &L, con);
	#endif

#if TIMER
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	cuda_stats.total_time += time;
	if (time > cuda_stats.max_time) 
	  cuda_stats.max_time = time;
	if (time < cuda_stats.min_time || cuda_stats.calls == 1) 
	  cuda_stats.min_time = time;
	hipEventDestroy(start);
	hipEventDestroy(stop);
	Cuda_Statistics();
#endif

	return res_rows;
}

