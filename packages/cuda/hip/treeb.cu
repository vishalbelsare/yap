#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/gather.h>
#include <thrust/scan.h>
#include <iostream>
#include <cstdio>
#include "lista.h"
#include "bpreds.h"

#define WARP_SIZE		32
#define TREE_NODE_SIZE		WARP_SIZE
#define TREE_FANOUT		(TREE_NODE_SIZE + 1)

#define N_MULTI_P 		16
#define BLCK_PER_MP_create	256	// blocks per multiprocessor during tree creation
#define BLCK_PER_MP_search	512	// blocks per multiprocessor during tree searching
#define WAPRS_PER_BLCK_join	8//16	// blocks per multiprocessor during tree creation
#define BLCK_PER_MP_join	512//256	// blocks per multiprocessor during tree searching

#define THRD_PER_BLCK_create	TREE_NODE_SIZE
#define BLCK_PER_GRID_create	(N_MULTI_P * BLCK_PER_MP_create)
#define THRD_PER_BLCK_search	TREE_NODE_SIZE
#define BLCK_PER_GRID_search	(N_MULTI_P * BLCK_PER_MP_search)
#define THRD_PER_GRID_search	(THRD_PER_BLCK_search * BLCK_PER_GRID_search)
#define THRD_PER_BLCK_join	(WARP_SIZE * WAPRS_PER_BLCK_join)
#define BLCK_PER_GRID_join	(N_MULTI_P * BLCK_PER_MP_join)
#define THRD_PER_GRID_join	(THRD_PER_BLCK_join * BLCK_PER_GRID_join)

#define TEST_MAX		100

typedef int IKeyType;
typedef int Record;

typedef struct {
	int keys[TREE_NODE_SIZE];
} IDirectoryNode;

typedef struct {
	Record records[TREE_NODE_SIZE];
} IDataNode;

typedef struct {
	IDataNode* data;
	unsigned int nDataNodes;
	IDirectoryNode* dir;
	unsigned int nDirNodes;
} CUDA_CSSTree;

struct to_neg
{
	__host__ __device__
	bool operator()(const int &r1)
	{
		if(r1 < 0)
			return 1;
		return 0;
	}
};

__host__ __device__ unsigned int uintCeilingLog(unsigned int base, unsigned int num)
{
	unsigned int result = 0;

	for(unsigned int temp = 1; temp < num; temp *= base)
		result++;

	return result;
}

__host__ __device__ unsigned int uintCeilingDiv(unsigned int dividend, unsigned int divisor)
{
	return (dividend + divisor - 1) / divisor;
}

__host__ __device__ unsigned int uintPower(unsigned int base, unsigned int pow)
{
	unsigned int result = 1;

	for(; pow; pow--)
		result *= base;

	return result;
}

__device__ int getRightMostDescIdx(int tree_size, int nodeIdx)
{
	int tmp = nodeIdx * TREE_NODE_SIZE + TREE_FANOUT;
	int n = uintCeilingLog(TREE_FANOUT, uintCeilingDiv(TREE_NODE_SIZE * tree_size + TREE_FANOUT, tmp)) - 1;

	int result = (tmp * uintPower(TREE_FANOUT, n) - TREE_FANOUT) / TREE_NODE_SIZE;
    	return result; 
}

__device__ int getDataArrayIdx(int dirSize, int tree_size, int bottom_start, int treeIdx)
{
	int idx;
	if(treeIdx < dirSize) {
		idx = tree_size - bottom_start - 1;
	}
	else if( treeIdx < bottom_start ) {
		idx = tree_size - bottom_start + treeIdx - dirSize;
	}
	else {
		idx = treeIdx - bottom_start;
	}
	return idx;
}

// Binary Search
__device__ int firstMatchingKeyInDirNode1(int keys[], int key)
{
	int min = 0;
	int max = TREE_NODE_SIZE;
	int mid;
	int cut;
	while(max - min > 1) {
		mid = (min + max) / 2;
		cut = keys[mid];

		if(key > cut)
			min = mid;
		else
			max = mid;
	}

	if(keys[min] >= key)
		return min;

	return max;

}

// Binary Search
__device__ int firstMatchingKeyInDataNode2(Record records[], IKeyType key)
{
	int min = 0;
	int max = TREE_NODE_SIZE;
	int mid;
	int cut;
	while(max - min > 1) {
		mid = (min + max) / 2;
		cut = records[mid];

		if(key > cut)
			min = mid;
		else
			max = mid;
	}

	if(records[min] == key)
		return min;

	if(max < TREE_NODE_SIZE && records[max] == key)
		return max;

	return -1;
}

__global__ void gCreateIndex(IDataNode data[], IDirectoryNode dir[], int dirSize, int tree_size, int bottom_start, int nNodesPerBlock)
{
        int startIdx = hipBlockIdx_x * nNodesPerBlock;
        int endIdx = startIdx + nNodesPerBlock;
        if(endIdx > dirSize)
                endIdx = dirSize;
        int keyIdx = hipThreadIdx_x;

        // Proceed only when in internal nodes
        for(int nodeIdx = startIdx; nodeIdx < endIdx; nodeIdx++)
        {
                int childIdx = nodeIdx * TREE_FANOUT + keyIdx + 1;        // One step down to the left
                // Then look for the right most descendent
                int rightMostDesIdx;
                // Common cases
                if(childIdx < tree_size) {
                        rightMostDesIdx = getRightMostDescIdx(tree_size, childIdx);
                }
                // versus the unusual case when the tree is incomplete and the node does not have the full set of children
                else {
                        // pick the last node in the tree (largest element of the array)`
                        rightMostDesIdx = tree_size - 1;
                }

                int dataArrayIdx = getDataArrayIdx(dirSize, tree_size, bottom_start, rightMostDesIdx);
	        dir[nodeIdx].keys[keyIdx] = data[dataArrayIdx].records[TREE_NODE_SIZE - 1];
        }
}

__global__ void gSearchTree(IDataNode* data, int nDataNodes, IDirectoryNode* dir, int nDirNodes, int lvlDir, Record* arr, int locations[], int nSearchKeys, int nKeysPerThread, int tree_size, int bottom_start)
{
	// Bringing the root node (visited by every tuple) to the faster shared memory
	__shared__ IKeyType RootNodeKeys[TREE_NODE_SIZE];
	RootNodeKeys[hipThreadIdx_x] = dir->keys[hipThreadIdx_x];

	__syncthreads();

	int OverallThreadIdx = hipBlockIdx_x * THRD_PER_BLCK_search + hipThreadIdx_x;

	for(int keyIdx = OverallThreadIdx; keyIdx < nSearchKeys; keyIdx += THRD_PER_GRID_search)
	{
		IKeyType val = arr[keyIdx];
		int loc = firstMatchingKeyInDirNode1(RootNodeKeys, val) + 1;
		for(int i = 1; i < lvlDir && loc < nDirNodes; i++) {
			int kid = firstMatchingKeyInDirNode1(dir[loc].keys, val);
			loc = loc * TREE_FANOUT + kid + 1;
		}

		if(loc >= tree_size)
			loc = nDataNodes - 1;
		else
			loc = getDataArrayIdx(nDirNodes, tree_size, bottom_start, loc);

		int offset = firstMatchingKeyInDataNode2(data[loc].records, val);
		locations[keyIdx] = (offset <0)?-1:(loc * TREE_NODE_SIZE + offset);
	}
}

/*Counts the number of times a row in 'S' is to be joined to a row in 'R'.*/
__global__ void gIndexJoin(int *R, int *S, int g_locations[], int sLen, int g_ResNums[])
{
	int s_cur = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;

	if(s_cur < sLen) 
	{
		int count = 1;
		int r_cur = g_locations[s_cur];
		int s_key;
		if(r_cur >= 0)
		{
			s_key = S[s_cur];
			r_cur++;
			while(s_key == R[r_cur]) 
			{
				count++;
				r_cur++;
			}
			g_ResNums[s_cur] = count;
		}
	}
}

/*Corrects 'gSearchTree' results when dealing with a negative multijoin. Uses the values found in 'g_locations' which indicate, for each row in 'R', if its going
to be joined (positive number) or not (-1). Works by checking the additional columns to be joined (i.e. all except the two used by 'gSearchTree') and changing to -1 
in 'g_locations' those rows that have equal values in the checked columns.*/
__global__ void gIndexMultiJoinNegative(int *R, int *S, int g_locations[], int rLen, int *p1, int *p2, int of1, int of2, int *mloc, int *sloc, int *muljoin, int wj)
{
	extern __shared__ int shared[];
	int r_cur = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
	int posr, poss, x;

	if(hipThreadIdx_x < wj)
		shared[hipThreadIdx_x] = muljoin[hipThreadIdx_x];
	__syncthreads();

	if(r_cur < rLen) 
	{
		int s_cur = g_locations[r_cur];
		int r_key;
		if(s_cur >= 0)
		{
			r_key = R[r_cur];
			if(mloc == NULL)
				posr = r_cur * of1;
			else
				posr = mloc[r_cur] * of1;
			while(r_key == S[s_cur])
			{
				poss = sloc[s_cur] * of2;
				for(x = 0; x < wj; x += 2)
				{
					if(p1[posr + shared[x]] != p2[poss + shared[x+1]])
						break;
				}
				if(x >= wj)
					return;
				s_cur++;
			}
			g_locations[r_cur] = -1;
		}
	}
}

/*Corrects 'gSearchTree' results when dealing with a multijoin. Uses the values found in 'g_locations' which indicate, for each row in 'S', if its going
to be joined (positive number) or not (-1). Works by checking the additional columns to be joined (i.e. all except the two used by 'gSearchTree') and counting the number of 
times a row in 'S' is to be joined to its corresponding row in 'R', storing the new result in 'g_locations'.*/
__global__ void gIndexMultiJoin(int *R, int *S, int g_locations[], int sLen, int g_ResNums[], int *p1, int *p2, int of1, int of2, int *mloc, int *sloc, int *muljoin, int wj)
{
	extern __shared__ int shared[];
	int s_cur = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
	int posr, poss, x;

	if(hipThreadIdx_x < wj)
		shared[hipThreadIdx_x] = muljoin[hipThreadIdx_x];
	__syncthreads();

	if(s_cur < sLen) 
	{
		int count = 0;
		int r_cur = g_locations[s_cur];
		int s_key;
		if(r_cur >= 0)
		{
			s_key = S[s_cur];
			if(sloc == NULL)
				poss = s_cur * of2;
			else
				poss = sloc[s_cur] * of2;
			while(s_key == R[r_cur]) 
			{
				posr = mloc[r_cur] * of1;
				for(x = 0; x < wj; x += 2)
				{
					if(p1[posr + shared[x]] != p2[poss + shared[x+1]])
						break;
				}
				if(x >= wj)
					count++;
				r_cur++;
			}
			if(count > 0)
				g_ResNums[s_cur] = count;
		}
	}
}

/*Writes the result of the join and projects the necessary columns as defined by 'rule'. The difference between this function and 'gJoinWithWrite' is the comparison of the additional join
columns.*/
__global__ void multiJoinWithWrite(int g_locations[], int sLen, int g_PrefixSums[], int g_joinResultBuffers[], int *p1, int *p2, int of1, int of2, int *rule, int halfrul, int lenrul, int *mloc, int *sloc, int wj)
{
	extern __shared__ int shared[];
	int *extjoins = &shared[lenrul];
	int s_cur = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;

	if(hipThreadIdx_x < (lenrul + wj))
		shared[hipThreadIdx_x] = rule[hipThreadIdx_x];
	__syncthreads();

	if(s_cur < sLen)
	{
		int r_cur = g_locations[s_cur];
		if(r_cur >= 0)
		{
			int x, y, pos, posr, poss;
			int num1 = g_PrefixSums[s_cur];
			int num2 = g_PrefixSums[s_cur+1];
			
			int tmp1, tmp2;

			if(sloc == NULL)	
				poss = s_cur * of2;
			else
				poss = sloc[s_cur] * of2;
			for(x = num1; x < num2; x++, r_cur++)
			{
				pos = mloc[r_cur] * of1;
				for(y = 0; y < wj; y += 2) /*Additional comparison*/
				{
					tmp1 = p1[pos + extjoins[y]];
					tmp2 = p2[poss + extjoins[y+1]];
					if(tmp1 != tmp2)
						break;
				}
				if(y < wj)
				{
					x--;
					continue;
				}
				posr = x * lenrul;
				for(y = 0; y < halfrul; y++)
					g_joinResultBuffers[posr + y] = p1[pos + shared[y]];
				for(; y < lenrul; y++)
					g_joinResultBuffers[posr + y] = p2[poss + shared[y]];
			}
		}
	}
}

/*Writes the result of the join and projects the necessary columns as defined by 'rule'. The difference between this function and 'gJoinWithWrite2' is the comparison of the additional join
columns.*/
__global__ void multiJoinWithWrite2(int g_locations[], int sLen, int g_PrefixSums[], int g_joinResultBuffers[], int *p1, int *p2, int of1, int of2, int *rule, int cols, int *mloc, int *sloc, int wj)
{
	extern __shared__ int shared[];
	int *extjoins = &shared[cols];
	int s_cur = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;

	if(hipThreadIdx_x < (cols + wj))
		shared[hipThreadIdx_x] = rule[hipThreadIdx_x];
	__syncthreads();

	if(s_cur < sLen)
	{
		int r_cur = g_locations[s_cur];
		if(r_cur >= 0)
		{
			int x, y, pos, pos2, posr, cond;
			int num1 = g_PrefixSums[s_cur];
			int num2 = g_PrefixSums[s_cur+1];
			if(sloc == NULL)
				pos2 = s_cur * of2 - 1;
			else
				pos2 = sloc[s_cur] * of2 - 1;
			for(x = num1; x < num2; x++, r_cur++)
			{
				pos = mloc[r_cur] * of1 - 1;
				for(y = 0; y < wj; y += 2) /*Additional comparison*/
				{
					if(p1[pos + extjoins[y] + 1] != p2[pos2 + extjoins[y+1] + 1])
						break;
				}
				if(y < wj)
				{
					x--;
					continue;
				}
				posr = x * cols;
				for(y = 0; y < cols; y++)
				{
					cond = shared[y];
					if(cond > 0)
						g_joinResultBuffers[posr + y] = p1[pos + cond];
					else
						g_joinResultBuffers[posr + y] = p2[pos2 - cond];
				}
			}
		}
	}
}

/*Writes the result of the join and projects the necessary columns as defined by 'rule'. The difference between this function and 'gJoinWithWrite2' is that only the columns in the positve
predicate are projected.*/
__global__ void gJoinWithWriteNegative(int g_locations[], int rLen, int g_joinResultBuffers[], int *p1, int of1, int *rule, int halfrul, int *mloc)
{
	extern __shared__ int shared[];
	int r_cur = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
	int posr;

	if(hipThreadIdx_x < halfrul)
		shared[hipThreadIdx_x] = rule[hipThreadIdx_x];
	__syncthreads();

	if(r_cur < rLen)
	{
		posr = g_locations[r_cur];
		if(g_locations[r_cur+1] != posr)
		{
			int y, pos;
			if(mloc == NULL)	
				pos = r_cur * of1;
			else
				pos = mloc[r_cur] * of1;
			posr *= halfrul;
			for(y = 0; y < halfrul; y++)
				g_joinResultBuffers[posr + y] = p1[pos + shared[y]];
		}
	}
}

/*Writes the result of the join and projects the necessary columns as defined by 'rule'. The difference between this function and 'gJoinWithWrite' is that only the columns in the positve
predicate are projected.*/
__global__ void gJoinWithWriteNegative2(int g_locations[], int rLen, int g_joinResultBuffers[], int *p1, int of1, int *rule, int cols, int *mloc)
{
	extern __shared__ int shared[];
	int r_cur = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
	int posr;

	if(hipThreadIdx_x < cols)
		shared[hipThreadIdx_x] = rule[hipThreadIdx_x];
	__syncthreads();

	if(r_cur < rLen)
	{
		posr = g_locations[r_cur];
		if(g_locations[r_cur+1] != posr)
		{
			int y, pos;
			if(mloc == NULL)
				pos = r_cur * of1 - 1;
			else
				pos = mloc[r_cur] * of1 - 1;
			posr *= cols;
			for(y = 0; y < cols; y++)
				g_joinResultBuffers[posr + y] = p1[pos + shared[y]];
		}
	}
}

/*Writes the result of the join and projects the necessary columns as defined by 'rule'.*/
__global__ void gJoinWithWrite(int g_locations[], int sLen, int g_PrefixSums[], int g_joinResultBuffers[], int *p1, int *p2, int of1, int of2, int *rule, int halfrul, int lenrul, int *mloc, int *sloc)
{
	extern __shared__ int shared[];
	int s_cur = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;

	if(hipThreadIdx_x < lenrul)
		shared[hipThreadIdx_x] = rule[hipThreadIdx_x];
	__syncthreads();

	if(s_cur < sLen)
	{
		int r_cur = g_locations[s_cur];
		if(r_cur >= 0)
		{
			int x, y, pos, posr, poss;
			int num1 = g_PrefixSums[s_cur];
			int num2 = g_PrefixSums[s_cur+1];
			if(sloc == NULL)	
				poss = s_cur * of2;
			else
				poss = sloc[s_cur] * of2;
			for(x = num1; x < num2; x++, r_cur++)
			{
				pos = mloc[r_cur] * of1;
				posr = x * lenrul;
				for(y = 0; y < halfrul; y++)
					g_joinResultBuffers[posr + y] = p1[pos + shared[y]];
				for(; y < lenrul; y++)
					g_joinResultBuffers[posr + y] = p2[poss + shared[y]];
			}
		}
	}
}

/*Writes the result of the join and projects the necessary columns as defined by 'rule'. This version is used when performing the final join of the rule and its only difference is the 
projection, which is performed based on the variables in the head of the rule.*/
__global__ void gJoinWithWrite2(int g_locations[], int sLen, int g_PrefixSums[], int g_joinResultBuffers[], int *p1, int *p2, int of1, int of2, int *rule, int cols, int *mloc, int *sloc)
{
	extern __shared__ int shared[];
	int s_cur = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;

	if(hipThreadIdx_x < cols)
		shared[hipThreadIdx_x] = rule[hipThreadIdx_x];
	__syncthreads();

	if(s_cur < sLen)
	{
		int r_cur = g_locations[s_cur];
		if(r_cur >= 0)
		{
			int x, y, pos, pos2, posr, cond;
			int num1 = g_PrefixSums[s_cur];
			int num2 = g_PrefixSums[s_cur+1];
			if(sloc == NULL)
				pos2 = s_cur * of2 - 1;
			else
				pos2 = sloc[s_cur] * of2 - 1;
			for(x = num1; x < num2; x++, r_cur++)
			{
				pos = mloc[r_cur] * of1 - 1;
				posr = x * cols;
				for(y = 0; y < cols; y++)
				{
					cond = shared[y];
					if(cond > 0)
						g_joinResultBuffers[posr + y] = p1[pos + cond];
					else
						g_joinResultBuffers[posr + y] = p2[pos2 - cond];
				}
			}
		}
	}
}

/*Load part of column 'wj' of 'p' in 'R'. Which values are loaded is defined by the prefix sum results in 'pos'.*/
__global__ void llenar(int *p, int *R, int len, int of, int wj, int *pos, int *ids)
{
	int id = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
	int cond;
	if(id < len)
	{
		cond = pos[id];
		if(pos[id+1] != cond)
		{
			R[cond] = p[id * of + wj];
			ids[cond] = id;
		}
	}
}

/*Load an entire column from 'p' into 'R'.*/
__global__ void llenarnosel(int *p, int *R, int len, int of, int wj)
{
	int id = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
	if(id < len)
		R[id] = p[id * of + wj];
}

__global__ void projectfinal(int *res, int rows, int cols, int *rule, int *out)
{
	extern __shared__ int shared[];
	int id = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;

	if(hipThreadIdx_x < cols)
		shared[hipThreadIdx_x] = rule[hipThreadIdx_x];
	__syncthreads();
	
	if(id < rows)
	{
		id *= cols;
		for(int y = 0; y < cols; y++)
			out[id + y] = res[id + shared[y]];
	}
}

void project(int *res, int resrows, int numcols1, int numcols2, int *proj, int **ret, int type)
{
	int z, *dcons, *d_Rout;
	int numthreads = 1024;
	//numthreads = 32;
	int blockllen = resrows / numthreads + 1;
	int sizepro = numcols2 * sizeof(int);
	reservar(&dcons, sizepro);
	if(type)
	{
		int *pt = (int *)malloc(sizepro);
		for(z = 0; z < numcols2; z++)
			pt[z] = proj[z] - 1;
		hipMemcpy(dcons, pt, sizepro, hipMemcpyHostToDevice); 
		//hipDeviceSynchronize(); //Small cudaMemcpys are asynchronous, uncomment this line if the pointer is being liberated before it is copied.
		free(pt);
	}
	else
		hipMemcpy(dcons, proj, sizepro, hipMemcpyHostToDevice);
	reservar(&d_Rout, resrows * sizepro);
	hipLaunchKernel(HIP_KERNEL_NAME(projectfinal), dim3(blockllen), dim3(numthreads), sizepro, 0, res, resrows, numcols1, dcons, d_Rout);
	hipFree(dcons);
	hipFree(*ret);
	*ret = d_Rout;
}

__global__ void projectadd(int *dop1, int *dop2, int rows1, int rows2, int cols1, int cols2, int *dhead, int hsize, int *res)
{
	extern __shared__ int shared[];
	int id = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
	int pos2, posr, x, y, cond;
	if(hipThreadIdx_x < hsize)
		shared[hipThreadIdx_x] = dhead[hipThreadIdx_x];
	__syncthreads();
	if(id < rows2)
	{
		posr = id * hsize * rows1;
		pos2 = id * cols2 - 1;
		for(x = 0; x < rows1; x++)
		{
			for(y = 0; y < hsize; y++)
			{
				cond = shared[y];
				if(cond > 0)
					res[posr + y] = dop1[cond-1];
				else
					res[posr + y] = dop2[pos2 - cond];
			}
			posr += hsize;
		}
	}
}

void juntar(int *dop1, int *dop2, int rows1, int rows2, int cols1, int cols2, int *proj, int pcols, int **ret)
{
	int sizepro, *dcons, *d_Rout;
	int numthreads = 1024;
	//numthreads = 32;
	int blockllen = rows2 / numthreads + 1;
	sizepro = pcols * sizeof(int);
	reservar(&dcons, sizepro);
	hipMemcpy(dcons, proj, sizepro, hipMemcpyHostToDevice);
	reservar(&d_Rout, rows1 * rows2 * sizepro);
	hipLaunchKernel(HIP_KERNEL_NAME(projectadd), dim3(blockllen), dim3(numthreads), sizepro, 0, dop1, dop2, rows1, rows2, cols1, cols2, dcons, pcols, d_Rout);
	hipFree(dcons);
	*ret = d_Rout;
}

/*Joins two predicates. Starts by performing all preliminary operations (selections, selfjoins, comparisons) on both predicates. Then a column pair is used to construct 
a CSS-Tree and that tree is searched for join positions. The positions are used in a prefix sum and its result allows us to write the result. Multijoins and negative 
predicates follow roughly the same process, but use different kernels.*/
int join(int *p1, int *p2, int rLen, int sLen, int of1, int of2, list<rulenode>::iterator rule, int pos, int bothops, int **ret, int ANDlogic)
{
	int pos2 = pos + 1;
	int *sel1 = NULL, nsel1 = 0;
	int *sel2 = rule->select[pos2];
	int nsel2 = rule->numsel[pos2];
	int *proj = rule->project[pos];
	int2 projp = rule->projpos[pos];
	int *sjoin1 = NULL, nsj1 = 0;
	int *sjoin2 = rule->selfjoin[pos2];
	int nsj2 = rule->numselfj[pos2];
	int *pred1 = NULL;
	int2 npred1 = make_int2(0,0);
	int *pred2 = rule->preds[pos2];
	int2 npred2 = rule->numpreds[pos2];
	int npred2tot = npred2.x + npred2.y;
	int *wherej = rule->wherejoin[pos];
	int numj = rule->numjoin[pos];
	int negative = rule->negatives[pos2+1];
	int flag;

	#ifdef ROCKIT
		ANDlogic = 0;
	#endif
	if(negative)
		ANDlogic = 1;

#if TIMER
	cuda_stats.joins++;
#endif

	int size, sizet, sizet2;
	if(bothops)
	{
		sel1 = rule->select[pos];
		nsel1 = rule->numsel[pos];
		sjoin1 = rule->selfjoin[pos];
		nsj1 = rule->numselfj[pos];
		pred1 = rule->preds[pos];
		npred1 = rule->numpreds[pos];
		sizet = maximo(10, of1, of2, nsel1, nsel2, projp.y + numj - 2, nsj1, nsj2, numj, npred1.x, npred2tot) * sizeof(int);
	}
	else
		sizet = maximo(7, of1, of2, nsel2, projp.y + numj - 2, nsj2, numj, npred2tot) * sizeof(int);
	
	int *dcons, *temp, *temp2 = NULL;
	int *d_R, *d_S;
	int blockllen, numthreads;
	
	int extraspace = TREE_NODE_SIZE - rLen % TREE_NODE_SIZE;
	int m32rLen = rLen + extraspace;
	int extraspaceS = TREE_NODE_SIZE - sLen % TREE_NODE_SIZE;
	int m32sLen = sLen + extraspaceS;
	if(m32rLen > m32sLen)
		sizet2 = (m32rLen + 1) * sizeof(int);
	else
		sizet2 = (m32sLen + 1) * sizeof(int);

	reservar(&dcons, sizet);
	reservar(&temp, sizet2);
	thrust::device_ptr<int> res = thrust::device_pointer_cast(temp);

	numthreads = 1024;
	//numthreads = 32;
	blockllen = sLen / numthreads + 1;
	int memSizeS, newLen = 0;
	int *posR = NULL, *posS = NULL;
	int sizem32S = 0, sizextra;

	#ifdef TIMER
	//cout << "INICIO" << endl;
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	#endif

	if(npred2.x > 0 || npred2.y > 0 || nsel2 > 0 || nsj2 > 0)
	{
		newLen = sLen + 1;
		hipMemsetAsync(temp, 0, newLen * sizeof(int));
	}

	if(npred2.x > 0 || npred2.y > 0)
	{
		size = npred2tot * sizeof(int);
		hipMemcpy(dcons, pred2, size, hipMemcpyHostToDevice);

		if(npred2.y > 0) /*Fix case when a(X,Y),b(Y,Z),Z > Y*/
		{
			reservar(&temp2, sizet2);
			hipMemsetAsync(temp2, 0, newLen * sizeof(int));
			//res = thrust::device_pointer_cast(temp2);
			hipLaunchKernel(HIP_KERNEL_NAME(bpreds), dim3(blockllen), dim3(numthreads), size, 0, p1, p2, sLen, of1, of2, dcons, npred2tot, npred2.x, temp + 1, temp2 + 1);
		}
		else
		{
			if(negative)
				hipLaunchKernel(HIP_KERNEL_NAME(bpreds), dim3(blockllen), dim3(numthreads), size, 0, p1, p2, sLen, of1, of2, dcons, npred2tot, npred2.x, temp + 1, NULL);
			else
				hipLaunchKernel(HIP_KERNEL_NAME(bpredsOR), dim3(blockllen), dim3(numthreads), size, 0, p1, p2, sLen, of1, of2, dcons, npred2tot, npred2.x, temp + 1, NULL);
		}

		if(nsel2 > 0)
		{
			size = nsel2 * sizeof(int);
			hipMemcpy(dcons, sel2, size, hipMemcpyHostToDevice);
			hipLaunchKernel(HIP_KERNEL_NAME(marcar), dim3(blockllen), dim3(numthreads), size, 0, p2, sLen, of2, dcons, nsel2, temp + 1);
		}

		if(nsj2 > 0)
		{
			size = nsj2 * sizeof(int);
			hipMemcpy(dcons, sjoin2, size, hipMemcpyHostToDevice);
			hipLaunchKernel(HIP_KERNEL_NAME(samejoin), dim3(blockllen), dim3(numthreads), size, 0, p2, sLen, of2, dcons, nsj2, temp + 1);
		}
	}
	else
	{
		if(nsel2 > 0)
		{
			size = nsel2 * sizeof(int);
			hipMemcpy(dcons, sel2, size, hipMemcpyHostToDevice);
			hipLaunchKernel(HIP_KERNEL_NAME(marcar2), dim3(blockllen), dim3(numthreads), size, 0, p2, sLen, of2, dcons, nsel2, temp + 1);

			if(nsj2 > 0)
			{
				size = nsj2 * sizeof(int);
				hipMemcpy(dcons, sjoin2, size, hipMemcpyHostToDevice);
				hipLaunchKernel(HIP_KERNEL_NAME(samejoin), dim3(blockllen), dim3(numthreads), size, 0, p2, sLen, of2, dcons, nsj2, temp + 1);
			}
		}
		else
		{
			if(nsj2 > 0)
			{
				size = nsj2 * sizeof(int);
				hipMemcpy(dcons, sjoin2, size, hipMemcpyHostToDevice);
				hipLaunchKernel(HIP_KERNEL_NAME(samejoin2), dim3(blockllen), dim3(numthreads), size, 0, p2, sLen, of2, dcons, nsj2, temp + 1);	
			}
			else
			{
				sizem32S = m32sLen * sizeof(int);
				reservar(&d_S, sizem32S);
				hipMemsetAsync(d_S + sLen, 0x7f, extraspaceS * sizeof(int));
				hipLaunchKernel(HIP_KERNEL_NAME(llenarnosel), dim3(blockllen), dim3(numthreads), 0, 0, p2, d_S, sLen, of2, wherej[1]);
			}
		}
	}
	
	if(npred2.x > 0 || npred2.y > 0 || nsel2 > 0 || nsj2 > 0)
	{
		flag = 0;
		while(flag != 1)
		{
			try
			{
				thrust::inclusive_scan(res + 1, res + newLen, res + 1);		
				flag = 1;
			}
			catch(std::bad_alloc &e)
			{
				limpiar("inclusive scan in join", 0);
			}				
		}	
		newLen = res[sLen];
	
		if(newLen == 0) // && !negative) ARREGLAR
		{
			hipFree(temp);
			hipFree(dcons);
			return 0;
		}

		extraspaceS = TREE_NODE_SIZE - newLen % TREE_NODE_SIZE;
		sizextra = extraspaceS * sizeof(int);
		m32sLen = newLen + extraspaceS;
		sizem32S = m32sLen * sizeof(int);

		reservar(&d_S, sizem32S);
		reservar(&posS, sizem32S);
		hipMemsetAsync(d_S + newLen, 0x7f, sizextra);
		hipMemsetAsync(posS + newLen, 0x7f, sizextra);
		hipLaunchKernel(HIP_KERNEL_NAME(llenar), dim3(blockllen), dim3(numthreads), 0, 0, p2, d_S, sLen, of2, wherej[1], temp, posS);
		sLen = newLen;
	}

	#ifdef TIMER
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	//cout << "Select1 = " << time << endl;
	cuda_stats.select1_time += time;

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	#endif

	blockllen = rLen / numthreads + 1;
	int sizem32;
	if(bothops)
	{
		if(temp2 != NULL)
		{
			hipFree(temp);
			temp = temp2;
			res = thrust::device_pointer_cast(temp);
			newLen = rLen + 1;
			if(nsel1 > 0)
			{
				size = nsel1 * sizeof(int);
				hipMemcpy(dcons, sel1, size, hipMemcpyHostToDevice);
				hipLaunchKernel(HIP_KERNEL_NAME(marcar), dim3(blockllen), dim3(numthreads), size, 0, p1, rLen, of1, dcons, nsel1, temp + 1);
			}
			if(nsj1 > 0)
			{
				size = nsj1 * sizeof(int);
				hipMemcpy(dcons, sjoin1, size, hipMemcpyHostToDevice);
				hipLaunchKernel(HIP_KERNEL_NAME(samejoin), dim3(blockllen), dim3(numthreads), size, 0, p1, rLen, of1, dcons, nsj1, temp + 1);
			}
			if(npred1.x > 0)
			{
				size = npred1.x * sizeof(int);
				hipMemcpy(dcons, pred1, size, hipMemcpyHostToDevice);
				if(ANDlogic)
					hipLaunchKernel(HIP_KERNEL_NAME(bpredsnormal), dim3(blockllen), dim3(numthreads), size, 0, p1, rLen, of1, dcons, npred1.x, temp + 1);
				else
					hipLaunchKernel(HIP_KERNEL_NAME(bpredsorlogic), dim3(blockllen), dim3(numthreads), size, 0, p1, rLen, of1, dcons, npred1.x, temp + 1);
			}
		}
		else
		{
			if(npred1.x > 0 || nsel1 > 0 || nsj1 > 0)
			{
				newLen = rLen + 1;
				hipMemsetAsync(temp, 0, newLen * sizeof(int));
			}

			if(nsel1 > 0)
			{
				size = nsel1 * sizeof(int);
				hipMemcpy(dcons, sel1, size, hipMemcpyHostToDevice);
				hipLaunchKernel(HIP_KERNEL_NAME(marcar2), dim3(blockllen), dim3(numthreads), size, 0, p1, rLen, of1, dcons, nsel1, temp + 1);

				if(nsj1 > 0)
				{
					size = nsj1 * sizeof(int);
					hipMemcpy(dcons, sjoin1, size, hipMemcpyHostToDevice);
					hipLaunchKernel(HIP_KERNEL_NAME(samejoin), dim3(blockllen), dim3(numthreads), size, 0, p1, rLen, of1, dcons, nsj1, temp + 1);
				}

				if(npred1.x > 0)
				{
					size = npred1.x * sizeof(int);
					hipMemcpy(dcons, pred1, size, hipMemcpyHostToDevice);
					if(ANDlogic)
						hipLaunchKernel(HIP_KERNEL_NAME(bpredsnormal), dim3(blockllen), dim3(numthreads), size, 0, p1, rLen, of1, dcons, npred1.x, temp + 1);
					else
						hipLaunchKernel(HIP_KERNEL_NAME(bpredsorlogic), dim3(blockllen), dim3(numthreads), size, 0, p1, rLen, of1, dcons, npred1.x, temp + 1);		
				}
			}
			else
			{
				if(nsj1 > 0)
				{
					size = nsj1 * sizeof(int);
					hipMemcpy(dcons, sjoin1, size, hipMemcpyHostToDevice);
					hipLaunchKernel(HIP_KERNEL_NAME(samejoin2), dim3(blockllen), dim3(numthreads), size, 0, p1, rLen, of1, dcons, nsj1, temp + 1);

					if(npred1.x > 0)
					{
						size = npred1.x * sizeof(int);
						hipMemcpy(dcons, pred1, size, hipMemcpyHostToDevice);
						if(ANDlogic)
							hipLaunchKernel(HIP_KERNEL_NAME(bpredsnormal), dim3(blockllen), dim3(numthreads), size, 0, p1, rLen, of1, dcons, npred1.x, temp + 1);
						else
							hipLaunchKernel(HIP_KERNEL_NAME(bpredsorlogic), dim3(blockllen), dim3(numthreads), size, 0, p1, rLen, of1, dcons, npred1.x, temp + 1);
					}
				}
				else
				{
					if(npred1.x > 0)
					{
						size = npred1.x * sizeof(int);
						hipMemcpy(dcons, pred1, size, hipMemcpyHostToDevice);
						if(ANDlogic)
							hipLaunchKernel(HIP_KERNEL_NAME(bpredsnormal2), dim3(blockllen), dim3(numthreads), size, 0, p1, rLen, of1, dcons, npred1.x, temp + 1);
						else
							hipLaunchKernel(HIP_KERNEL_NAME(bpredsorlogic2), dim3(blockllen), dim3(numthreads), size, 0, p1, rLen, of1, dcons, npred1.x, temp + 1);	
					}
				}
			}
		}

		if(temp2 != NULL || npred1.x > 0 || nsel1 > 0 || nsj1 > 0)
		{
			thrust::inclusive_scan(res + 1, res + newLen, res + 1);
			newLen = res[rLen];
			if(newLen == 0)
			{
				hipFree(temp);
				hipFree(dcons);
				hipFree(d_S);
				if(posS != NULL)
					hipFree(posS);
				return 0;
			}	

			extraspace = TREE_NODE_SIZE - newLen % TREE_NODE_SIZE;
			sizextra = extraspace * sizeof(int);
			m32rLen = newLen + extraspace;
			sizem32 = m32rLen * sizeof(int);

			reservar(&d_R, sizem32);
			reservar(&posR, sizem32);
			hipMemsetAsync(d_R + newLen, 0x7f, sizextra);
			hipMemsetAsync(posR + newLen, 0x7f, sizextra);
			hipLaunchKernel(HIP_KERNEL_NAME(llenar), dim3(blockllen), dim3(numthreads), 0, 0, p1, d_R, rLen, of1, wherej[0], temp, posR);
			rLen = newLen;
		}
		else
		{
			sizem32 = m32rLen * sizeof(int);
			reservar(&d_R, sizem32);
			hipMemsetAsync(d_R + rLen, 0x7f, extraspace * sizeof(int));
			hipLaunchKernel(HIP_KERNEL_NAME(llenarnosel), dim3(blockllen), dim3(numthreads), 0, 0, p1, d_R, rLen, of1, wherej[0]);
		}
	}
	else
	{
		sizem32 = m32rLen * sizeof(int);
		reservar(&d_R, sizem32);
		hipMemsetAsync(d_R + rLen, 0x7f, extraspace * sizeof(int));
		hipLaunchKernel(HIP_KERNEL_NAME(llenarnosel), dim3(blockllen), dim3(numthreads), 0, 0, p1, d_R, rLen, of1, wherej[0]);
	}

	#ifdef TIMER
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	//cout << "Select2 = " << time << endl;
	cuda_stats.select2_time += time;
	#endif

	#ifdef TIMER
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	#endif

	thrust::device_ptr<Record> dvp1;
	thrust::device_ptr<Record> permutation;
	if(negative)
	{
		dvp1 = thrust::device_pointer_cast(d_S);
		if(posS == NULL)
		{
			reservar(&posS, sizem32S);
			permutation = thrust::device_pointer_cast(posS);
			thrust::sequence(permutation, permutation + m32sLen);
		}
		else
			permutation = thrust::device_pointer_cast(posS);

		flag = 0;
		while(flag != 1)
		{
			try
			{
				thrust::stable_sort_by_key(dvp1, dvp1 + m32sLen, permutation);
				flag = 1;
			}
			catch(std::bad_alloc &e)
			{
				limpiar("inclusive scan in join", 0);
			}
		}
	}
	else
	{
		dvp1 = thrust::device_pointer_cast(d_R);
		if(posR == NULL)
		{
			reservar(&posR, sizem32);
			permutation = thrust::device_pointer_cast(posR);
			thrust::sequence(permutation, permutation + m32rLen);
		}
		else
			permutation = thrust::device_pointer_cast(posR);

		flag = 0;
		while(flag != 1)
		{
			try
			{
				thrust::stable_sort_by_key(dvp1, dvp1 + m32rLen, permutation);
				flag = 1;
			}
			catch(std::bad_alloc &e)
			{
				limpiar("inclusive scan in join", 0);
			}
		}
	}

	#ifdef TIMER
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	//cout << "Sort = " << time << endl;
	cuda_stats.sort_time += time;
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	#endif

	IDataNode* d_data;
	IDirectoryNode* d_dir;
	unsigned int nDataNodes;

	if(negative)
	{
		nDataNodes = uintCeilingDiv(sLen, TREE_NODE_SIZE);
		d_data=(IDataNode *)d_S;
	}
	else
	{
		nDataNodes = uintCeilingDiv(rLen, TREE_NODE_SIZE);
		d_data=(IDataNode *)d_R;
	}

	unsigned int lvlDir = uintCeilingLog(TREE_FANOUT, nDataNodes);
	unsigned int nDirNodes = uintCeilingDiv(nDataNodes - 1, TREE_NODE_SIZE);
	unsigned int tree_size = nDirNodes + nDataNodes;
	unsigned int bottom_start = (uintPower(TREE_FANOUT, lvlDir) - 1) / TREE_NODE_SIZE;
	d_dir = (IDirectoryNode *)temp;

	unsigned int nNodesPerBlock = uintCeilingDiv(nDirNodes, BLCK_PER_GRID_create);

	dim3 Dbc(THRD_PER_BLCK_create, 1, 1);
	dim3 Dgc(BLCK_PER_GRID_create, 1, 1);

	hipLaunchKernel(HIP_KERNEL_NAME(gCreateIndex), dim3(Dgc), dim3(Dbc), 0, 0, d_data, d_dir, nDirNodes, tree_size, bottom_start, nNodesPerBlock);

	int *d_locations;
	int memSizeR;
	unsigned int nSearchKeys;
	if(negative)
	{
		memSizeR = (rLen + 1) * sizeof(int);
		reservar(&d_locations, memSizeR);
		hipMemsetAsync(d_locations, 0, sizeof(int));
		nSearchKeys = rLen;
	}
	else
	{
		memSizeS = sLen * sizeof(int);
		reservar(&d_locations, memSizeS);
		nSearchKeys = sLen;
	}
	dim3 Dbs(THRD_PER_BLCK_search, 1, 1);
	dim3 Dgs(BLCK_PER_GRID_search, 1, 1);
	unsigned int nKeysPerThread = uintCeilingDiv(nSearchKeys, THRD_PER_GRID_search);
	if(negative)
	{
		hipLaunchKernel(HIP_KERNEL_NAME(gSearchTree), dim3(Dgs), dim3(Dbs), 0, 0, d_data, nDataNodes, d_dir, nDirNodes, lvlDir, d_R, d_locations + 1, nSearchKeys, nKeysPerThread, tree_size, bottom_start);
		hipMemsetAsync(temp, 0, memSizeR);
	}
	else
	{
		hipLaunchKernel(HIP_KERNEL_NAME(gSearchTree), dim3(Dgs), dim3(Dbs), 0, 0, d_data, nDataNodes, d_dir, nDirNodes, lvlDir, d_S, d_locations, nSearchKeys, nKeysPerThread, tree_size, bottom_start);
		hipMemsetAsync(temp, 0, memSizeS);
	}

	int muljoin = 0, muljoinsize = 0, sum;
	int *d_Rout;
	int resSize, sizepro;
	if(negative)
	{
		blockllen = rLen / numthreads + 1;
		if(numj > 2)
		{
			muljoin = numj - 2;
			muljoinsize = muljoin * sizeof(int);
			hipMemcpy(dcons, wherej + 2, muljoinsize, hipMemcpyHostToDevice);
			hipLaunchKernel(HIP_KERNEL_NAME(gIndexMultiJoinNegative), dim3(blockllen), dim3(numthreads), muljoinsize, 0, d_R, d_S, d_locations + 1, rLen, p1, p2, of1, of2, posR, posS, dcons, muljoin);
		}

		res = thrust::device_pointer_cast(d_locations);	
		thrust::transform(res + 1, res + rLen + 1, res + 1, to_neg());
		thrust::inclusive_scan(res + 1, res + rLen + 1, res + 1);
		sum = res[rLen];

		if(pos == (rule->num_rows - 3))
		{
			sizepro = rule->num_columns * sizeof(int);
			hipMemcpy(dcons, proj, sizepro, hipMemcpyHostToDevice);
			resSize = sum * sizepro;
			reservar(&d_Rout, resSize);
			hipLaunchKernel(HIP_KERNEL_NAME(gJoinWithWriteNegative2), dim3(blockllen), dim3(numthreads), sizepro, 0, d_locations, rLen, d_Rout, p1, of1, dcons, rule->num_columns, posR);
		}
		else
		{	
			sizepro = projp.x * sizeof(int);
			hipMemcpy(dcons, proj, sizepro, hipMemcpyHostToDevice);
			resSize = sum * sizepro;
			reservar(&d_Rout, resSize);
			hipLaunchKernel(HIP_KERNEL_NAME(gJoinWithWriteNegative), dim3(blockllen), dim3(numthreads), sizepro, 0, d_locations, rLen, d_Rout, p1, of1, dcons, projp.x, posR);
		}
		hipFree(d_R);
		hipFree(d_S);
	}
	else
	{
		blockllen = sLen / numthreads + 1;
		if(numj > 2)
		{
			muljoin = numj - 2;
			muljoinsize = muljoin * sizeof(int);
			hipMemcpy(dcons, wherej + 2, muljoinsize, hipMemcpyHostToDevice);
			hipLaunchKernel(HIP_KERNEL_NAME(gIndexMultiJoin), dim3(blockllen), dim3(numthreads), muljoinsize, 0, d_R, d_S, d_locations, sLen, temp, p1, p2, of1, of2, posR, posS, dcons, muljoin);
		}
		else
			hipLaunchKernel(HIP_KERNEL_NAME(gIndexJoin), dim3(blockllen), dim3(numthreads), 0, 0, d_R, d_S, d_locations, sLen, temp);
		hipFree(d_R);
		hipFree(d_S);

		sum = res[sLen-1];
		thrust::exclusive_scan(res, res + sLen, res);
		sum += res[sLen-1];
		if(sum == 0)
		{
			hipFree(dcons);
			hipFree(d_locations);
			hipFree(temp);
			if(posS != NULL)
				hipFree(posS);
			if(posR != NULL)
				hipFree(posR);
			return 0;
		}
		res[sLen] = sum;

		if(pos == (rule->num_rows - 3))
		{
			sizepro = rule->num_columns * sizeof(int);
			hipMemcpy(dcons, proj, sizepro, hipMemcpyHostToDevice);
			resSize = sum * sizepro;
			reservar(&d_Rout, resSize);
			if(numj > 2)
			{
				hipMemcpy(dcons + rule->num_columns, wherej + 2, muljoinsize, hipMemcpyHostToDevice);
				hipLaunchKernel(HIP_KERNEL_NAME(multiJoinWithWrite2), dim3(blockllen), dim3(numthreads), sizepro + muljoinsize, 0, d_locations, sLen, temp, d_Rout, p1, p2, of1, of2, dcons, rule->num_columns, posR, posS, muljoin);
			}
			else
				hipLaunchKernel(HIP_KERNEL_NAME(gJoinWithWrite2), dim3(blockllen), dim3(numthreads), sizepro, 0, d_locations, sLen, temp, d_Rout, p1, p2, of1, of2, dcons, rule->num_columns, posR, posS);
		}
		else
		{
			sizepro = projp.y * sizeof(int);
			hipMemcpy(dcons, proj, sizepro, hipMemcpyHostToDevice);
			resSize = sum * sizepro;
			reservar(&d_Rout, resSize);
			if(numj > 2)
			{
				hipMemcpy(dcons + projp.y, wherej + 2, muljoinsize, hipMemcpyHostToDevice);
				hipLaunchKernel(HIP_KERNEL_NAME(multiJoinWithWrite), dim3(blockllen), dim3(numthreads), sizepro + muljoinsize, 0, d_locations, sLen, temp, d_Rout, p1, p2, of1, of2, dcons, projp.x, projp.y, posR, posS, muljoin);
			}
			else
				hipLaunchKernel(HIP_KERNEL_NAME(gJoinWithWrite), dim3(blockllen), dim3(numthreads), sizepro, 0, d_locations, sLen, temp, d_Rout, p1, p2, of1, of2, dcons, projp.x, projp.y, posR, posS);
		}
	}

	hipFree(dcons);
	hipFree(d_locations);
	hipFree(temp);
	if(posS != NULL)
		hipFree(posS);
	if(posR != NULL)
		hipFree(posR);
	
	if(*ret != NULL)
		hipFree(*ret);
	*ret = d_Rout;

	#ifdef TIMER
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	//cout << "Join = " << time << endl;
	//cout << "FIN" << endl;
	cuda_stats.join_time += time;
	#endif

	return sum;
}
