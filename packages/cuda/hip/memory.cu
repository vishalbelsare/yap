#include <hip/hip_runtime.h>
#include <list>
#include <iostream>
#include <stdlib.h>
#include <algorithm>
#include <thrust/device_vector.h>
#include "lista.h"
#include "memory.h"
#include "pred.h"

#define MAX_REC 200
#define MAX_FIX_POINTS 100

memnode temp_storage[MAX_REC];
/*List used to store information (address, size, etc.) about facts and rule results loaded in the GPU*/
list<memnode> GPUmem;
/*List used to store information about rule results offloaded from the GPU to the CPU*/
list<memnode> CPUmem;

/*Auxiliary function to sort rule list*/
bool comparer(const rulenode &r1, const rulenode &r2)
{
	return (r1.name > r2.name); 
}

/*Used in search functions to compare iterations*/
bool compareiteration(const memnode &r1, const memnode &r2)
{
	return (r1.iteration < r2.iteration); 
}

/*Used in search functions to compare names*/
bool comparename(const memnode &r1, const memnode &r2)
{
	return (r1.name > r2.name); 
}

/*Linear search of 'name' fact*/
template<class InputIterator>
InputIterator buscarhecho(InputIterator first, InputIterator last, int name)
{
	while(first!=last) 
	{
		if(first->name == name && first->isrule == 0) return first;
			++first;
	}
	return last;
}

/*Finds all results of rule 'name' in iteration 'itr' in both CPU and GPU memory. Every result found is removed from its respective list*/
list<memnode>::iterator buscarpornombre(int name, int itr, int *totalrows, int *gpunum, int *cpunum)
{
	int x = 0, sum = 0;
	memnode temp;
	list<memnode>::iterator i;
	temp.iteration = itr;
	pair<list<memnode>::iterator, list<memnode>::iterator> rec = equal_range(GPUmem.begin(), GPUmem.end(), temp, compareiteration);

	while(rec.first != rec.second)
	{
		if(rec.first->name == name && rec.first->isrule == 1)
		{
			temp_storage[x] = *rec.first;
			rec.first = GPUmem.erase(rec.first);
			sum += temp_storage[x].rows;
			x++;
		}	
		else
			rec.first++;
	}
	*gpunum = x;
	temp.name = name;
	temp.isrule = 1;
	i = GPUmem.insert(rec.first, temp);
	rec = equal_range(CPUmem.begin(), CPUmem.end(), temp, compareiteration);

	while(rec.first != rec.second)
	{				
		if(rec.first->name == name && rec.first->isrule == 1)
		{
			temp_storage[x] = *rec.first;
			rec.first = CPUmem.erase(rec.first);
			sum += temp_storage[x].rows;
			x++;
		}	
		else
			rec.first++;
	}
	*totalrows = sum;
	*cpunum = x;
	return i;
}

list<memnode>::iterator buscarpornombrecpu(int name, int itr, int *totalrows, int *gpunum, int *cpunum)
{
	int x = 0, sum = 0;
	memnode temp;
	list<memnode>::iterator i;
	temp.iteration = itr;
	pair<list<memnode>::iterator, list<memnode>::iterator> rec = equal_range(GPUmem.begin(), GPUmem.end(), temp, compareiteration);

	while(rec.first != rec.second)
	{				
		if(rec.first->name == name)
		{
			temp_storage[x] = *rec.first;
			rec.first = GPUmem.erase(rec.first);
			sum += temp_storage[x].rows;
			x++;
		}	
		else
			rec.first++;
	}

	*gpunum = x;
	temp.name = name;
	temp.isrule = 1;
	rec = equal_range(CPUmem.begin(), CPUmem.end(), temp, compareiteration);

	while(rec.first != rec.second)
	{				
		if(rec.first->name == name)
		{
			temp_storage[x] = *rec.first;
			rec.first = CPUmem.erase(rec.first);
			sum += temp_storage[x].rows;
			x++;
		}	
		else
			rec.first++;
	}
	i = CPUmem.insert(rec.first, temp);
	*totalrows = sum;
	*cpunum = x;
	return i;
}

/*Removes the least recently used memory block from GPU memory, sending it to CPU memory if it's a rule result. 
If there are no used memory blocks in the GPU and we still don't have enough memory, the program exits with error*/
void limpiar(const char s[], size_t sz)
{
	list<memnode>::iterator ini;
	memnode temp;
	size_t free, total;

	if(GPUmem.size() == 0)
	{
		hipMemGetInfo(&free,&total);
		cerr << s << ": not enough GPU memory: have " << free << " of " << total << ", need " << sz << " bytes." << endl;
		exit(1);
	}		

	ini = GPUmem.begin();
	if(ini->isrule)
	{	
		temp = *ini;
		temp.dev_address = (int *)malloc(ini->size);
		hipMemcpyAsync(temp.dev_address, ini->dev_address, temp.size, hipMemcpyDeviceToHost);
		list<memnode>::iterator pos = lower_bound(CPUmem.begin(), CPUmem.end(), temp, compareiteration);
		CPUmem.insert(pos, temp);
	}
	hipFree(ini->dev_address);
	GPUmem.erase(ini);
}

/*Allocs 'size' amount of bytes in GPU memory. If not enough memory is available, removes least recently used memory blocks until 
enough space is available*/
void reservar(int **ptr, size_t size)
{
	size_t free, total;

        if (size == 0) { 
                *ptr = NULL; 
                return;
        }

	hipMemGetInfo(&free, &total);
	while(free < size)
	{
		cout << "Se limpio memoria " << free << " " << total << endl;
		limpiar("not enough memory", size);
		hipMemGetInfo(&free, &total);
	}

	while(hipMalloc(ptr, size) == hipErrorMemoryAllocation)
		limpiar("Error in memory allocation", size);
	if (! *ptr ) {
	  size_t free, total;
	  hipMemGetInfo(      &free, &total	 );
	  cerr << "Could not allocate " << size << " bytes, only " << free << " avaliable from total of " << total << " !!!" << endl;
	  cerr << "Exiting CUDA...." << endl;
	  exit(1);
	}
}

/*Creates a new entry in the GPU memory list*/
void registrar(int name, int num_columns, int *ptr, int rows, int itr, int rule)
{
	memnode temp;
	temp.name = name;
	temp.dev_address = ptr;
	temp.rows = rows;
	temp.size = rows * num_columns * sizeof(int);
	temp.iteration = itr;
	temp.isrule = rule;
	GPUmem.push_back(temp);
}

void registrarcpu(int name, int num_columns, int *ptr, int rows, int itr, int rule)
{
	memnode temp;
	temp.name = name;
	temp.dev_address = ptr;
	temp.rows = rows;
	temp.size = rows * num_columns * sizeof(int);
	temp.iteration = itr;
	temp.isrule = rule;
	CPUmem.push_back(temp);
}

/*Updates the information of an element in a list*/
template<class InputIterator>
void actualizar(int num_columns, int *ptr, int rows, InputIterator i)
{
	i->dev_address = ptr;
	i->rows = rows;
	i->size = rows * num_columns * sizeof(int);
}

/*Count the total number of rows generated by rule 'name' in iteration 'iter'*/
int numrows(int name, int itr)
{
	int sum = 0;
	memnode temp;
	temp.iteration = itr;
	pair<list<memnode>::iterator, list<memnode>::iterator> rec = equal_range(GPUmem.begin(), GPUmem.end(), temp, compareiteration);
	while(rec.first != rec.second)
	{
		if(rec.first->name == name)
			sum += rec.first->rows;
		rec.first++;
	}
	rec = equal_range(CPUmem.begin(), CPUmem.end(), temp, compareiteration);
	while(rec.first != rec.second)
	{
		if(rec.first->name == name)
			sum += rec.first->rows;
		rec.first++;
	}
	return sum;
}

	extern "C" void * YAP_IntToAtom(int);
	extern  "C" char * YAP_AtomName(void *);

/*Loads facts or rule results in GPU memory. If a fact is already in GPU memory, its pointer is simply returned. Otherwise, 
memory is reserved and the fact is loaded. Rule results are loaded based on the current iteration 'itr' and both GPU and 
CPU memories are searched for all instances of said results. The instances are combined into a single one in GPU memory.*/
int cargar(int name, int num_rows, int num_columns, int is_fact, int *address_host_table, int **ptr, int itr)
{
	int numgpu, numcpu, totalrows = 0;
	int *temp, x;
	int size, itrant, inc = 0;
	list<memnode>::iterator i;
	memnode fact;

	if(is_fact)
	{
		i = buscarhecho(GPUmem.begin(), GPUmem.end(), name);
		if(i != GPUmem.end())
		{
			fact = *i;
			GPUmem.erase(i);
			fact.iteration = itr;
			*ptr = fact.dev_address;
			GPUmem.push_back(fact);
			return fact.rows;
		}
		size = num_rows * num_columns * sizeof(int);
		reservar(&temp, size);
		hipMemcpyAsync(temp, address_host_table, size, hipMemcpyHostToDevice);
		registrar(name, num_columns, temp, num_rows, itr, 0);
		*ptr = temp;
		return num_rows;
	}
	if(itr > 0)
	{
		itrant = itr - 1;
		i = buscarpornombre(name, itrant, &totalrows, &numgpu, &numcpu);
		if((numgpu == 1) && (numcpu == 1))
		{
			actualizar(num_columns, temp_storage[0].dev_address, temp_storage[0].rows, i);
			*ptr = temp_storage[0].dev_address;
			return temp_storage[0].rows;
		}
		size = totalrows * num_columns * sizeof(int);
		reservar(&temp, size);
		for(x = 0; x < numgpu; x++)
		{
			hipMemcpyAsync(temp + inc, temp_storage[x].dev_address, temp_storage[x].size, hipMemcpyDeviceToDevice);
			inc += temp_storage[x].size / sizeof(int);
			hipFree(temp_storage[x].dev_address);
		}
		for(; x < numcpu; x++)
		{
			hipMemcpyAsync(temp + inc, temp_storage[x].dev_address, temp_storage[x].size, hipMemcpyHostToDevice);
			inc += temp_storage[x].size / sizeof(int);
			free(temp_storage[x].dev_address);
		}
		actualizar(num_columns, temp, totalrows, i);
		*ptr = temp;
		return totalrows;
	}
	return 0;
}

int cargarcpu(int name, int num_rows, int num_columns, int is_fact, int *address_host_table, int **ptr, int itr)
{
	int numgpu, numcpu, totalrows = 0;
	int *temp, x;
	int size, itrant, inc = 0;
	list<memnode>::iterator i;

	if(is_fact)
	{
		*ptr = address_host_table;
		return num_rows;
	}
	if(itr > 0)
	{
		itrant = itr - 1;
		i = buscarpornombrecpu(name, itrant, &totalrows, &numgpu, &numcpu);

		if((numgpu == 0) && (numcpu == 1))
		{
			actualizar(num_columns, temp_storage[0].dev_address, temp_storage[0].rows, i);
			*ptr = temp_storage[0].dev_address;
			return temp_storage[0].rows;
		}
		size = totalrows * num_columns * sizeof(int);
		temp = (int *)malloc(size);
		for(x = 0; x < numgpu; x++)
		{
			hipMemcpyAsync(temp + inc, temp_storage[x].dev_address, temp_storage[x].size, hipMemcpyDeviceToHost);
			inc += temp_storage[x].size / sizeof(int);
			hipFree(temp_storage[x].dev_address);
		}
		for(; x < numcpu; x++)
		{
			memcpy(temp + inc, temp_storage[x].dev_address, temp_storage[x].size);
			inc += temp_storage[x].size / sizeof(int);
			free(temp_storage[x].dev_address);
		}
		actualizar(num_columns, temp, totalrows, i);
		*ptr = temp;
		return totalrows;
	}
	return 0;
}

/*Loads all results of rule 'name' from both GPU and CPU memories into the GPU*/
int cargafinal(int name, int cols, int **ptr)
{
	int *temp, *ini, cont = 0, numg = 0, numc = 0;
	memnode bus;
	bus.name = name;
	GPUmem.sort(comparename);
	CPUmem.sort(comparename);
	list<memnode>::iterator endg = GPUmem.end();
	list<memnode>::iterator endc = CPUmem.end();
	list<memnode>::iterator pos = lower_bound(GPUmem.begin(), endg, bus, comparename);
	list<memnode>::iterator gpu = pos;
	while(pos != endg && pos->name == name)
	{
		cont += pos->rows;
		numg++;
		pos++;
	}
	pos = lower_bound(CPUmem.begin(), endc, bus, comparename);
	list<memnode>::iterator cpu = pos;
	while(pos != endc && pos->name == name)
	{
		cont += pos->rows;
		numc++;
		pos++;
	}

	if(numg == 0 && numc == 0)
		return 0;
	if(numg == 1 && numc == 0) 
	{
		pos = gpu;
		*ptr = pos->dev_address;
		cont = pos->rows;
		GPUmem.erase(pos);
		#ifdef TUFFY
		return -cont;
		#else
		return cont;
		#endif
	}
	if(numg == 0 && numc == 1)
	{
		pos = cpu;
		cont = pos->rows;
		#ifdef TUFFY
		reservar(&temp, pos->size);
		hipMemcpy(temp, pos->dev_address, pos->size, hipMemcpyHostToDevice);
		*ptr = temp;
		#else
		*ptr = pos->dev_address;
		#endif
		CPUmem.erase(pos);
		return -cont;
	}

	reservar(&temp, cont * cols * sizeof(int));
	ini = temp;
	pos = gpu;
	while(pos != endg && pos->name == name)
	{
		hipMemcpy(temp, pos->dev_address, pos->size, hipMemcpyDeviceToDevice);
		temp += pos->size / sizeof(int);
		pos++;
	}
	pos = cpu;
	while(pos != endc && pos->name == name)
	{
		hipMemcpy(temp, pos->dev_address, pos->size, hipMemcpyHostToDevice);
		temp += pos->size / sizeof(int);
		pos++;
	}
	*ptr = ini;
	return cont;
}

/*Compares the results of the current iteration against the results of older iterations. 
Used to avoid infinite computations when the result is not a single fixed-point, but an 
orbit of points.*/
bool generadas(int name, int filas, int cols, int itr)
{
	int r1, r2, x, fin;
	int *dop1, *dop2;

	r2 = numrows(name, itr);
	if(itr < MAX_FIX_POINTS)
		fin = itr;
	else
		fin = MAX_FIX_POINTS;
	for(x = 1; x <= fin; x++)
	{
		r1 = numrows(name, itr - x);
		if(r1 == r2)
		{
			r2 = cargar(name, filas, cols, 0, NULL, &dop2, itr + 1);
			thrust::device_ptr<int> pt2 = thrust::device_pointer_cast(dop2);
			r1 = cargar(name, filas, cols, 0, NULL, &dop1, itr - x + 1);
			thrust::device_ptr<int> pt1 = thrust::device_pointer_cast(dop1);
			if(thrust::equal(pt1, pt1 + r1, pt2) == true)
				return true;
		}
	}
	return false;
}

void mostrar_memoria()
{
	unsigned int x;
	list<memnode>::iterator i = GPUmem.begin();
	cout << "Memoria inicio GPU" << endl;
	for(x = 0; x < GPUmem.size(); x++, i++)
		cout << i->name << " " << i->iteration << " " << i->isrule << " " << i->rows << " " << i->size << endl;
	cout << "Memoria fin GPU" << endl;
}

void mostrar_memcpu()
{
	unsigned int x;
	list<memnode>::iterator i = CPUmem.begin();
	cout << "Memoria inicio CPU" << endl;
	for(x = 0; x < CPUmem.size(); x++, i++)
		cout << i->name << " " << i->iteration << endl;
	cout << "Memoria fin CPU" << endl;
}

/*Clear all rule results from both GPU and CPU memory*/
void clear_memory()
{
	list<memnode>::iterator ini;
	list<memnode>::iterator fin;
       	ini = GPUmem.begin();
	fin = GPUmem.end();
	while(ini != fin)
	{
		if(ini->isrule)
		{
			hipFree(ini->dev_address);
			ini = GPUmem.erase(ini);
		}
		else
			ini++;
	}
	ini = CPUmem.begin();
	fin = CPUmem.end();
	while(ini != fin)
	{
		free(ini->dev_address);
		ini++;
	}
	CPUmem.clear();
}

/*Clear everything from both GPU and CPU memory*/
void clear_memory_all()
{
	list<memnode>::iterator ini;
	list<memnode>::iterator fin;
       	ini = GPUmem.begin();
	fin = GPUmem.end();
	while(ini != fin)
	{
		hipFree(ini->dev_address);
		ini++;
	}
	GPUmem.clear();
	ini = CPUmem.begin();
	fin = CPUmem.end();
	while(ini != fin)
	{
		free(ini->dev_address);
		ini++;
	}
	CPUmem.clear();
}

/*Remove all instances of fact 'name' from both CPU and GPU memories*/
void liberar(int name)
{
	list<memnode>::iterator i;
	memnode fact;
	i = buscarhecho(GPUmem.begin(), GPUmem.end(), name);
	if(i != GPUmem.end())
	{
		fact = *i;
		GPUmem.erase(i);
		hipFree(fact.dev_address);
	}
	i = buscarhecho(CPUmem.begin(), CPUmem.end(), name);
	if(i != CPUmem.end())
	{
		fact = *i;
		CPUmem.erase(i);
		free(fact.dev_address);
	}
}

/*Add all rows in 'dop1' to the fact 'name' by creating a new array capable of holding both.*/
void sumar(int name, int *dop1, int cols, int rows)
{
	list<memnode>::iterator i;
	memnode fact;
	i = buscarhecho(GPUmem.begin(), GPUmem.end(), name);
	int *res, newrows, offset;
	if(i != GPUmem.end())
	{
		fact = *i;
		newrows = rows + fact.rows;
		reservar(&res, newrows * cols * sizeof(int));
		offset = fact.rows * cols;
		hipMemcpyAsync(res, fact.dev_address, offset * sizeof(int), hipMemcpyDeviceToDevice);
		GPUmem.erase(i);
		registrar(name, cols, res, newrows, 0, 0);
		hipMemcpyAsync(res + offset, dop1, rows * cols * sizeof(int), hipMemcpyDeviceToDevice);
		hipFree(fact.dev_address);
	}
}
